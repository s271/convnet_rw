#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>

#include <layer_kernels.cuh>

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];  
        labelLogProbs[tx] = __logf(labelp);
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

#define L1_SVM_C .3

__global__ void kL2SVMCost(float* acts, float* labels, float* maxActs, float* acts_out, float* correctPreds,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);

        const float max_svm = maxActs[tx];
        const float svm_label_value = acts[label * numCases + tx]; 

        if (svm_label_value != max_svm) {
            correctPreds[tx] = 0;

        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += acts[i * numCases + tx] == max_svm;
            }
            correctPreds[tx] = 1.0f / float(numMax);
        }

		float sum_svm = 0;
		for(int i = 0; i < numOut; i++)
		{
			const float wx = acts[i * numCases + tx];
			float svm_val;
			 if (wx != max_svm)
				 svm_val =  fmaxf(1+max_svm, 0);
			 else
				 svm_val =  fmaxf(1-max_svm, 0);
			 sum_svm +=  svm_val*svm_val + L1_SVM_C*svm_val;
		}

        acts_out[tx] = sum_svm;
    }
}



__global__ void kRLogCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
						  float* probWeights, const float p_pow, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];  
		float logprob = __logf(labelp);
        labelLogProbs[tx] = logprob;
		//float err =  fmaxf(__logf(maxp) - logprob, 0);
		float w = __powf(-logprob + 1e-6, p_pow);//*.6/(.6 + err);

		probWeights[tx] = w;
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dy_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregCostGrad(float* y_l, float* labels, float* dE_dy_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * (label == ty);
        v = __fdividef(v, y_l[tidx]);
        if (add) {
            dE_dy_l[tidx] += v;
        } else {
            dE_dy_l[tidx] = v;
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dy_l: (numOut, numCases)
 */
template <bool add>
__global__ void kRLogCostGrad(float* y_l, float* labels, float* dE_dy_l, float* weights, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
		const float w = weights[tx];
        float v = w*gradCoeff * (label == ty);
        v = __fdividef(v, y_l[tidx]);
        if (add) {
            dE_dy_l[tidx] += v;
        } else {
            dE_dy_l[tidx] = v;
        }
    }
}

/*
 * dE_dy_l: (numOut, numCases)
 * y_l:     (numOut, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        float v = 0;
        for (int j = 0; j < numOut; j++) {
            v += dE_dy_l[j * numCases + tx] * ((j == ty) - y_l[j * numCases + tx]);
        }
        v *= y_l[tidx];
        
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}
__device__ inline float Psvm(float a, float invCp1, float C2) {
	return (a<0)*a + (a>C2)*invCp1*(a-C2);
};

__device__ inline float Gradsvm(float a, float C1, float C2) {
	return C1*fmax(a, 0) + C2*(a > 0);
};

__device__ inline float GradPsvm(float a, float invCp1z, float Cz, float C1, float C2) {
	return (a>Cz)*(C2 + C1*invCp1z*(a-Cz));
};

__global__ void kL2SVM_G(float* racts, float* acts, float* labels, float* sumZ2, float* G, const int numCases,
                                 const int numOut, const float C1, const float C2, const float eps_w, const float eps_b) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
	
//eps = 1/lambda
   
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
		float t = (label == ty)?1:-1;
		float val = (1 - t*(racts[tidx]+acts[tidx]));

		const float ZL = sumZ2[tx]*eps_w + eps_b;

		const float invCp1z = 1.f/(1 + C1*ZL);
		const float Cz = C2*ZL;

		G[tidx] = t*GradPsvm(val, invCp1z, Cz, C1, C2);
    }

}

__global__ void kL2SVM_U(float* acts, float* labels, float* actU, const int numCases,
                                 const int numOut, const float C1eps, const float C2eps) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
	
//eps = 1/lambda
   
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
		float t = (label == ty)?1:-1;
		float val = (1 - t*acts[tidx]);

		const float invCp1 = 1.f/(1 + C1eps);

		actU[tidx] = Psvm(val, invCp1, C2eps);
    }

}


template <bool add>
__global__ void kL2SVMGrad(float* y_l, float* labels, float* dE_dx_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
		float t = (label == ty)?1:-1;
		//y_l = w*act_prev
        //float v = gradCoeff*t*(1 - t*y_l[tidx] > 0); //-grad, because we are adding it and minimize
		float act = 1 - t*y_l[tidx];
		float max_val = fmaxf(act, 0) + L1_SVM_C*(act > 0);

		float v = gradCoeff*t*max_val; //-grad, because we are adding it and minimize
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregSoftmaxGrad(float* y_l, float* labels, float* dE_dx_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * ((label == ty) - y_l[tidx]);
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

template <bool add>
__global__ void kRLogSoftmaxGrad(float* y_l, float* labels, float* dE_dx_l, float* probWeights, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);

		float p =  y_l[tidx];
		float w = probWeights[tx];

        float v = gradCoeff * ((label == ty) - p)*w;
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}


void computeL2SVMCost(NVMatrix& labels, NVMatrix& act_prev, NVMatrix& act_out, NVMatrix& correctPreds_out)
{
    int numCases = act_prev.getNumCols(); 
    int numOut = act_prev.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!act_prev.isTrans());
    assert(labels.isContiguous());
    assert(act_prev.isContiguous());
    
    NVMatrix& maxActs = act_prev.max(0);
    
    act_out.resize(1, numCases);
    correctPreds_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kL2SVMCost), hipFuncCachePreferL1);

    kL2SVMCost<<<blocks, threads>>>(act_prev.getDevData(), labels.getDevData(), maxActs.getDevData(),
                                    act_out.getDevData(), correctPreds_out.getDevData(),
                                    numCases, numOut);
    cutilCheckMsg("computeL2SVMCost: Kernel execution failed");

    delete &maxActs;
};

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
void computeLogregCost(NVMatrix& labels, NVMatrix& probs, NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
                                     numCases, numOut);
    cutilCheckMsg("computeLogregCost: Kernel execution failed");
//    hipDeviceSynchronize();
    delete &maxProbs;
}

void computeLogregGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregCostGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregCostGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeLogregGrad: Kernel execution failed");
}

void computeRLogCost(NVMatrix& labels, NVMatrix& probs,
					 NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out, NVMatrix& probWeights_out,
					 float p_pow) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRLogCost), hipFuncCachePreferL1);
    kRLogCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
									 probWeights_out.getDevData(), p_pow, numCases, numOut);
    cutilCheckMsg("computeRLogCost: Kernel execution failed");

    delete &maxProbs;
}

void computeRLogGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, NVMatrix& probWeights, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kRLogCostGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(), probWeights.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kRLogCostGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(), probWeights.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeLogregGrad: Kernel execution failed");
}

void computeL2SVMGrad(NVMatrix& labels, NVMatrix& acts, NVMatrix& target, bool add, float coeff)
{
    int numCases = acts.getLeadingDim(); 
    int numOut = acts.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(acts.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(acts.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(acts);
        kL2SVMGrad<false><<<blocks, threads>>>(acts.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kL2SVMGrad<true><<<blocks, threads>>>(acts.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

};

void computeSoftmaxGrad(NVMatrix& acts, NVMatrix& actsGrad, NVMatrix& target, bool add) {
    int numCases = acts.getLeadingDim();
    int numOut = acts.getFollowingDim();

    assert(acts.isSameDims(actsGrad));
    assert(acts.isContiguous());
    assert(actsGrad.isContiguous());
    assert(target.isContiguous());
    assert(acts.isTrans());
    assert(actsGrad.isTrans());

    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(acts);
        kSoftmaxGrad<false><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    } else {
        kSoftmaxGrad<true><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    }
    cutilCheckMsg("computeSoftmaxGrad: Kernel execution failed");
}

void computeLogregSoftmaxGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregSoftmaxGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregSoftmaxGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeLogregSoftmaxGrad: Kernel execution failed");
}

void computeRLogSoftmaxGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, NVMatrix& probWeights, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(probs.isTrans());

	if(!labels.isSameDims(probWeights)) {
		printf("computeRLogSoftmaxGrad - probWeights dimesions are wrong! \n");
		exit(EXIT_FAILURE);
	}
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kRLogSoftmaxGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(), probWeights.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kRLogSoftmaxGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(), probWeights.getDevData(),
                                                     numCases, numOut, coeff);
    }

    cutilCheckMsg("computeRLogSoftmaxGrad: Kernel execution failed");
}