#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>

#include <layer_kernels.cuh>
//-------------------------------------------------------------
//EltwiseMax
//-------------------------------------------------------------
template <int B_X, bool add>
__global__ void kEltwiseMaxGrad(float* actGrad, float* input, float* output, float* target,
                                const int numElements) {
    for (int i = B_X * blockIdx.x + threadIdx.x; i < numElements; i += B_X * gridDim.x) {
        if (add) {
            target[i] += actGrad[i] * (output[i] == input[i]);
        } else {
            target[i] = actGrad[i] * (output[i] == input[i]);
        }
    }
}

#include "tt.h"

#define CONST_AREA_SIZE 256
__device__ __constant__ float const_area[CONST_AREA_SIZE];

//-------------------------------------------------------------
//EltwiseFunc
//-------------------------------------------------------------
template <int sizeArr>
__global__ void kEltwiseFuncAct(const float* input, float* const target,
								const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideTag,
								const uint sizeIn, const uint sizeOut) {

	const int numPixelsPerGroup = imgInPixels/sizeIn;	

//    dim3 blocks(std::min(NUM_BLOCKS_MAX, DIVUP(out_width, ELTWISE_THREADS_X)),
//                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

// ix, iy == 0 almost always
    for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) {

        for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x) {	
			
			float inpVal[sizeArr];//use shared instead?
#pragma unroll
			for (uint inp_i = 0; inp_i < sizeIn; inp_i++) {	
				Offset inpOffset;
				inpOffset << Index(inp_i)
				<< numPixelsPerGroup
				<< Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
				<< strideInp
				<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

				float val = input[inpOffset._offset];
				inpVal[inp_i] = val;
			}
#pragma unroll		
			for (uint out_i = 0; out_i < sizeOut; out_i++) {
				int out_par = out_i*sizeIn*2;

				float output = 0;
#pragma unroll			
				for (uint inp_i = 0; inp_i < sizeIn; inp_i++)
				{		
					float param = const_area[out_par + inp_i];
					float paramM = const_area[out_par + sizeIn + inp_i];
					float val = inpVal[inp_i];
					output += param*val + paramM*fmax(val, 0);
				}// inp_i

				Offset tagOffset;
				tagOffset << Index(out_i)
				<< numPixelsPerGroup
				<< Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
				<< strideTag
				<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);
				target[tagOffset._offset] = output;
			}//out_i
        }
    }


}

template <int B_X, int B_Y, int sizeArr>
__global__ void kEltwiseFuncAct_t(const float* input, float* const target,
								const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideTag,
								const uint sizeIn, const uint sizeOut) {

	const int numPixelsPerGroup = imgInPixels/sizeIn;	

//    dim3 blocks(std::min(NUM_BLOCKS_MAX, DIVUP(out_width, ELTWISE_THREADS_X)),
//                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

// ix, iy == 0 almost always

//go over output group
    const uint idxX = blockIdx.x * B_X + threadIdx.x;

//go over cases
    const uint idxY = blockIdx.y * B_Y + threadIdx.y;

	//gridDim.y is DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)
    //for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y * B_Y)
	{

        //for (uint ix = 0; ix < numCases; ix += gridDim.x * B_X)
		{	

		float inpVal[sizeArr];//use shared instead?

			uint x = idxX;// + ix;
			uint yg = idxY;// + iy;

			for (uint inp_i = 0; inp_i < sizeIn; inp_i++) {	
				int yt = yg + inp_i*numPixelsPerGroup;
				float val = input[yt * strideInp + x];
				inpVal[inp_i] = val;
			}		
	
			for (uint out_i = 0; out_i < sizeOut; out_i++) {
				int out_par = out_i*sizeIn*2;

				float output = 0;
			
				for (uint inp_i = 0; inp_i < sizeIn; inp_i++)
				{		
					float param = const_area[out_par + inp_i];
					float paramM = const_area[out_par + sizeIn + inp_i];
					float val = inpVal[inp_i];
					output += param*val + paramM*fmax(val, 0);
				}// inp_i

				int yo = yg;//+ out_i*numPixelsPerGroup;
				int offseTag = yo * strideTag + x;
				target[offseTag] = output;
			}//out_i

        }
    }
}

template <int sizeArr>
__global__ void kEltwiseFuncGrad(const float* actGrad, const float* input, float* const target,
								const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideOut,
								const uint sizeIn, const uint sizeOut) {


	const int numPixelsPerGroup = imgInPixels/sizeIn;	
	const int inStep = strideInp*numPixelsPerGroup;
	const int outStep = strideOut*numPixelsPerGroup;
//with no N_SUM ix, iy == 0 almost always
    for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) {
        for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x) {	

			float grad_next[sizeArr];

			Offset offset;
			offset 
			<< Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
			<< strideInp
			<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

			for (uint out_i = 0; out_i < sizeOut; out_i++)
			{
				grad_next[out_i] = actGrad[offset._offset + outStep*out_i];
			}//out_i

			for (uint inp_i = 0; inp_i < sizeIn; inp_i++) {	
				int inp_offset = offset._offset + inp_i*inStep;

				float val = input[inp_offset];
				float vsign = (val > 0);
				float sum_grad = 0;
				
				for (uint out_i = 0; out_i < sizeOut; out_i++)	
					sum_grad += grad_next[out_i]
					*(vsign*const_area[out_i*sizeIn*2 + sizeIn + inp_i]
						+ const_area[out_i*sizeIn*2 + inp_i]);

				target[inp_offset] = sum_grad;
			}	

		}//ix
	}//iy

}

template <int B_X, int B_Y, int sizeArr>
__global__ void kEltwiseFuncGrad_t(const float* actGrad, const float* input, float* const target,
								const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideOut,
								const uint sizeIn, const uint sizeOut) {

//go over output group
    const uint idxX = blockIdx.x * B_X + threadIdx.x;
//go over cases
    const uint idxY = blockIdx.y * B_Y + threadIdx.y;

	const int numPixelsPerGroup = imgInPixels/sizeIn;
//	const int inStep = strideInp*numPixelsPerGroup;
//	const int outStep = strideOut*numPixelsPerGroup;

	//gridDim.y is DIVUP(numOutPixelsPerGroup, ELTWISE_THREADS_Y)
    for (uint iy = idxY; iy < numPixelsPerGroup; iy += gridDim.y * B_Y) {

        for (uint ix = idxX; ix < numCases; ix += gridDim.x * B_X) {

			float grad_next[sizeArr];

			for (uint out_i = 0; out_i < sizeOut; out_i++)
				grad_next[out_i] = actGrad[(iy + out_i*numPixelsPerGroup)*strideOut + ix];

			for (uint inp_i = 0; inp_i < sizeIn; inp_i++) {	
				int yt = iy + inp_i*numPixelsPerGroup;
				int offset = yt * strideInp + ix;
				float val = input[offset];
				float vsign = (val > 0);
				float sum_grad = 0;
				
				for (uint out_i = 0; out_i < sizeOut; out_i++)	
					sum_grad += grad_next[out_i]*(vsign*const_area[out_i*sizeIn*2 + sizeIn + inp_i]
						+ const_area[out_i*sizeIn*2 + inp_i]); //optimize away later

				target[offset] = sum_grad;

			}	

			//int offset = y * stride + x;

			//float in0 = input0[offset];
			//float in1 = input1[offset];
			//float in2 = input2[offset];
			//float grad_next = actGrad[offset];

			//float val0 = param0 + param3*(in0 > 0);
			//float val1 = param1 + param4*(in1 > 0);
			//float val2 = param2 + param5*(in2 > 0);

			//target0[offset] = val0*grad_next;
			//target1[offset] = val1*grad_next;
			//target2[offset] = val2*grad_next;

   //         //float val = param0*in0 + param1*in1 + param2*in2 + param3*fm0 + param4*fm1 + param5*fm2;

		}
   }

}

__global__ void kEltwiseFuncParamGradSingle(float* actGrad, float* input, float* target, float* target_m,
								const uint pin, const uint pout, const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideOut, const uint strideTag,
								const uint sizeIn, const uint sizeOut)
{
	const int numPixelsPerGroup = imgInPixels/sizeIn;	


	float sum = 0;
	float sum_m = 0;

#pragma unroll	
    for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) {
#pragma unroll
      for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x) {	

			Offset offsetInp;
			offsetInp
			<< Index(pin)
			<< numPixelsPerGroup
			<< Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
			<< strideInp
			<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);
			
			float in_val = input[offsetInp._offset];

			Offset offsetOut;
			offsetOut
			<< Index(pout)
			<< numPixelsPerGroup
			<< Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
			<< strideOut
			<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

			float grad_next = actGrad[offsetOut._offset];

			float val_m = fmax(in_val, 0);
			sum += grad_next*in_val;
			sum_m += grad_next*val_m;
		}
	}

	Offset offsetTag;
	offsetTag
	<< Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
	<< strideTag
	<< Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

	target[offsetTag._offset] = sum;
	target_m[offsetTag._offset] = sum_m;

}


template <int B_X, int B_Y>
__global__ void kEltwiseFuncParamGradSingle_t(float* actGrad, float* input, float* target, float* target_m,
								const uint pin, const uint pout, const uint imgInPixels, const uint numCases,
								const uint strideInp, const uint strideOut, const uint strideTag,
								const uint sizeIn, const uint sizeOut)
{
	const int numPixelsPerGroup = imgInPixels/sizeIn;	
    const uint idxX = blockIdx.x * B_X + threadIdx.x;
    const uint idxY = blockIdx.y * B_Y + threadIdx.y;

	float sum = 0;
	float sum_m = 0;

#pragma unroll
    for (uint y = idxY; y < numPixelsPerGroup; y += gridDim.y * B_Y) {
#pragma unroll
        for (uint x = idxX; x < numCases; x += gridDim.x * B_X) {
			int offset = y * strideInp + x;
			float in_val = input[offset + pin*numPixelsPerGroup* strideInp ];

			float grad_next = actGrad[y * strideOut + x + pout*numPixelsPerGroup* strideInp ];

			float val_m = fmax(in_val, 0);
			sum += grad_next*in_val;
			sum_m += grad_next*val_m;
		}
	}
	int tagOffset = (threadIdx.x + blockIdx.x*blockDim.x) +  (threadIdx.y + blockIdx.y*blockDim.y)*strideTag;

	target[tagOffset] = sum;
	target_m[tagOffset] = sum_m;

}
//-------------------------------------------------------------
//MicroConv
//-------------------------------------------------------------
#define SMEM(X, Y, sdata) sdata[(X)*sharedY+(Y) + sOffset]

#define SHARED_MEM(x, y, z, LOBE, getVal, sdata) \
    SMEM((LOBE) + sx, (LOBE) + sy, sdata) = getVal(x, y, z);\
    if (sx < (LOBE)) {\
        SMEM(sx, (LOBE) + sy, sdata) = getVal(max(x - (LOBE), 0), y, z);\
        SMEM((LOBE) + bw + sx, (LOBE) + sy, sdata) = getVal(min(x + bw, imgSizeX-1), y, z);\
    }\
    if (sy < (LOBE)) {\
        SMEM((LOBE) + sx, sy, sdata) = getVal(x, max(y - (LOBE), 0), z);\
        SMEM((LOBE) + sx, (LOBE) + bh + sy, sdata) = getVal(x, min(y + bh, imgSizeY-1), z);\
    }\
    if ((sx < (LOBE)) && (sy < (LOBE))) {\
        SMEM(sx, sy, sdata) = getVal(max(x - (LOBE), 0), max(y - (LOBE), 0), z);\
        SMEM(sx, (LOBE) + bh + sy, sdata) = getVal(max(x - (LOBE), 0), min(y + bh, imgSizeY-1), z);\
        SMEM((LOBE) + bw + sx, sy, sdata) = getVal(min(x + bw, imgSizeX-1), max(y - (LOBE), 0), z);\
        SMEM((LOBE) + bw + sx, (LOBE) + bh + sy, sdata) = getVal(min(x + bw, imgSizeX-1), min(y + bh, imgSizeY-1), z);\
    }

#define getValInput(X, Y, Z) input[channelOffset + (X)*widthyz+(Y)*widthz + (Z)]

template < int LOBE, int SIZE_CONV>
__global__ void kMicroConvFilterAct(const float* input, float* const target,
								const uint numCases, const uint channels, const uint numFilters, const uint casePerThread,
								const uint sharedY, const uint modulesPerBlockX,  const uint modulesPerBlockY, 
								const uint imgSizeX, const uint imgSizeY,
								const uint imgPixels)
{
	extern __shared__ float sdata[];
//order x>y>z, *not* y>x
	const int bsizeX = imgSizeX/modulesPerBlockX;
	const int bsizeY = imgSizeY/modulesPerBlockY;
	const int startX = (blockIdx.y/bsizeY)*modulesPerBlockX;
	const int startY = (blockIdx.y%bsizeY)*modulesPerBlockY;

    const int  bw = modulesPerBlockX;
    const int  bh = modulesPerBlockY;
    const int  sx = threadIdx.y/modulesPerBlockY;
    const int  sy = threadIdx.y - sx*modulesPerBlockY;

	const int  ix = sx+startX;
	const int  iy = sy+startY;

	const int widthz = numCases;
	const int widthyz = imgSizeY*numCases;

	const int sizeConv2 = SIZE_CONV*SIZE_CONV;
	const int sharedY2 = sharedY*sharedY;


//put pragme unroll here	
	for(int zind = 0; zind < casePerThread; zind++)
	{
		const int z = threadIdx.x + blockIdx.x*blockDim.x + zind*blockDim.x*gridDim.x;			
		for(int channelInd = 0; channelInd < channels; channelInd++)
		{	
			const int sOffset = channelInd*sharedY2*blockDim.x + threadIdx.x*sharedY2;
			const int channelOffset = channelInd*imgPixels*numCases;

			if(z < numCases)
			{

				SHARED_MEM(ix, iy, z, LOBE, getValInput, sdata)	
			}
		}

		__syncthreads();

		for(int channelInd = 0; channelInd < channels; channelInd++)
		{	
			const int sOffset = channelInd*sharedY2*blockDim.x + threadIdx.x*sharedY2;
			const int channelOffset = channelInd*imgPixels*numCases;

			if(z < numCases)
			{
				for(int filterID = 0; filterID <  numFilters; filterID++)
				{
						float sum = 0;

						for(int dsx = - LOBE; dsx < LOBE+1; dsx++)
						for(int dsy = - LOBE; dsy <  LOBE+1; dsy++)
						{
							int idx = min(max(ix + dsx, 0), imgSizeX-1);
							int idy = min(max(iy + dsy, 0), imgSizeY-1);

							float sd = sdata[(sx + dsx + LOBE)*sharedY+(sy + dsy + LOBE) + sOffset];

							sum += sd*const_area[channelInd*sizeConv2*numFilters + filterID*sizeConv2 + (dsy + LOBE)*SIZE_CONV +(dsx + LOBE)];
						}
									
						target[numFilters*channelOffset + filterID*imgPixels*numCases + ix*widthyz + iy*widthz + z] = sum;

				}//filter
			}//if
		}//channel
	}//zind
}
#define getValAct(X, Y, Z) actGrad[filterOffset + (X)*widthyz+(Y)*widthz + (Z)]

__global__ void kMicroConvActGrad(const float* actGrad, float* const target,
								const uint numCases, const uint channels, const uint numFilters, const uint casePerThread,
								const uint modulesPerBlockX, const uint modulesPerBlockY,
								const uint sharedY, const uint sizeModule, const uint lobe,
								const uint imgSizeX, const uint imgSizeY,
								const uint imgPixels)
{
	extern __shared__ float sdata[];
//order x>y>z, *not* y>x
	
	const int bsizeX = imgSizeX/modulesPerBlockX;
	const int bsizeY = imgSizeY/modulesPerBlockY;
	const int startX = (blockIdx.y/bsizeY)*modulesPerBlockX;
	const int startY = (blockIdx.y%bsizeY)*modulesPerBlockY;

    const int  bw = modulesPerBlockX;
    const int  bh = modulesPerBlockY;
    const int  sx = threadIdx.y/modulesPerBlockY;
    const int  sy = threadIdx.y - sx*modulesPerBlockY;

	const int  ix = sx+startX;
	const int  iy = sy+startY;

	const int widthz = numCases;
	const int widthyz = imgSizeY*numCases;

	const int sizeModule2 = sizeModule*sizeModule;
	const int sharedY2 = sharedY*sharedY;

	for(int zind = 0; zind < casePerThread; zind++)
	{
		const int z = threadIdx.x + blockIdx.x*blockDim.x + zind*blockDim.x*gridDim.x;		
	//pragma unroll here

		for(int channelInd = 0; channelInd < channels; channelInd++)
		{
			const int channelOffset = channelInd*imgPixels*numCases;

			float sum = 0;
			for(int filterID = 0; filterID <  numFilters; filterID++)
			{
				const int sOffset = channelInd*numFilters*sharedY2*blockDim.x + filterID*sharedY2*blockDim.x + threadIdx.x*sharedY2;
				const int filterOffset = numFilters*channelOffset + filterID*imgPixels*numCases;

				SHARED_MEM(ix, iy, z, lobe, getValAct, sdata)	
			}
		}

		__syncthreads();

		for(int channelInd = 0; channelInd < channels; channelInd++)
		{
			const int channelOffset = channelInd*imgPixels*numCases;

			float sum = 0;
			for(int filterID = 0; filterID <  numFilters; filterID++)
			{
				const int sOffset = channelInd*numFilters*sharedY2*blockDim.x + filterID*sharedY2*blockDim.x + threadIdx.x*sharedY2;
				const int filterOffset = numFilters*channelOffset + filterID*imgPixels*numCases;
				
				for(int dsx = - lobe; dsx < lobe+1; dsx++)
				for(int dsy = - lobe; dsy <  lobe+1; dsy++)
					sum += sdata[(sx + dsx + lobe)*sharedY+(sy + dsy + lobe)]
							*const_area[filterID*sizeModule2 + (-dsy + lobe)*sizeModule +(-dsx + lobe)];

			}
			target[channelOffset + ix*widthyz + iy*widthz + z] = sum;
		}
	}
}

template <int lobe>
__global__ void kMicroConvWeightGrad(const float* actGrad, const float* input, float** const target,
								const uint target_size, const uint numCases, const uint casePerThread, const uint tagWidth,
								const uint channels, const uint numFilters, 
								const uint modulesPerBlockX, const uint modulesPerBlockY,
								const uint imgSizeX, const uint imgSizeY, const uint imgPixels)
{

//order x>y>z, *not* y>x
	extern __shared__ float sdata[];
	const int imgSize = imgSizeX*imgSizeY;
	const int sharedY = modulesPerBlockY + 2*lobe;
	const int sizeSharedBlock = sharedY*(modulesPerBlockX + 2*lobe);
	float* sdataImg = sdata;
	float* sdataRes = sdata + sizeSharedBlock*blockDim.x;

	const int bsizeX = imgSizeX/modulesPerBlockX;
	const int bsizeY = imgSizeY/modulesPerBlockY;
	const int startX = (blockIdx.y/bsizeY)*modulesPerBlockX;
	const int startY = (blockIdx.y%bsizeY)*modulesPerBlockY;

    const int  bw = modulesPerBlockX;
    const int  bh = modulesPerBlockY;
    const int  sx = threadIdx.y/modulesPerBlockY;
    const int  sy = threadIdx.y - sx*modulesPerBlockY;

	const int  ix = sx+startX;
	const int  iy = sy+startY;

	const int zoff = threadIdx.x + blockIdx.x*blockDim.x;

	const int widthz = numCases;
	const int widthyz = imgSizeY*numCases;

	const int sharedY2 = sharedY*sharedY;

	const int conv_size = 2*lobe+1;
	const int conv2 = conv_size*conv_size;

	int resStride = numFilters*conv2;
	int res_off = resStride*(threadIdx.y*blockDim.x + threadIdx.x);

	const int sOffset = threadIdx.x*sharedY2;

	for(int channelInd = 0; channelInd < channels; channelInd++)
	{
		const int channelOffset = channelInd*imgPixels*numCases;

		memset(sdataRes + res_off, 0, resStride*sizeof(float));

		for(int zind = 0; zind < casePerThread; zind++)
		{

			const int z = zoff + zind*blockDim.x*gridDim.x;		
			for(int filterID = 0; filterID <  numFilters; filterID++)
			{

				SHARED_MEM(ix, iy, z, lobe, getValInput, sdataImg)	

				__syncthreads();

				for(int dsx = - lobe; dsx < lobe+1; dsx++)
				for(int dsy = - lobe; dsy < lobe+1; dsy++)
				{
					int idx = min(max(ix + dsx, 0), imgSizeX-1);
					int idy = min(max(iy + dsy, 0), imgSizeY-1);

					const int filterOffset = numFilters*channelOffset + filterID*imgPixels*numCases;				
					float vact = actGrad[filterOffset + ix*widthyz + iy*widthz + z];
					float vimg = sdataImg[(sx + dsx + lobe)*sharedY+(sy + dsy + lobe) + sOffset];
						//input[channelOffset + idx*widthyz + idy*widthz + z];

					int ind_coeff = filterID*conv2 + (dsy + lobe)*conv_size +(dsx + lobe);
					sdataRes[res_off + ind_coeff] += vact*vimg;


				}//dsx
			}//filter

		}//z

		for(int isx = 0; isx < conv_size; isx++)
		for(int isy = 0; isy < conv_size; isy++)
		{
			for(int filterID = 0; filterID <  numFilters; filterID++)
			{
				int ind_coeff = filterID*conv2 + isy*conv_size + isx;
				int ind_ch = ind_coeff + channelInd*numFilters*conv2;
				target[ind_ch][ix*imgSizeX*tagWidth + tagWidth*iy + zoff] = sdataRes[res_off + ind_coeff];
			}
		}

	}//channel

}

//-------------------------------------------------------------
//VectFunc
//-------------------------------------------------------------
#define SCALE_H 1.

template <int sizeV>
__global__ void kVectFuncAct(const float* input, float* const target,
								const uint numPixelsPerGroup, const uint numCases,
								const uint strideInp, const uint strideTag, int numColors, int sizeH) {

// ix, iy == 0 almost always
	const int bd_off =  (blockDim.y*blockIdx.y + threadIdx.y)*strideInp + blockDim.x*blockIdx.x + threadIdx.x;
	const int pix_stride = numPixelsPerGroup*strideInp;
	const int pix_tag_stride = numPixelsPerGroup*strideTag;

    for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) 
	{
        for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x)
		{	

			int xy_off = iy*strideInp +	ix + bd_off;

			for (uint color = 0; color < numColors; color ++) {	

				int color_off =  color*pix_stride;
			
				float inpVal[sizeV];//use shared instead?
	#pragma unroll
				for (uint inp_i = 0; inp_i < sizeV; inp_i++) {					

					int voff = color_off*sizeV + inp_i*pix_stride + xy_off;

					float val = input[voff];

					inpVal[inp_i] = val;
				}

				float vmax= 0;
#pragma unroll	
				for (uint out_i = 0; out_i < sizeH; out_i++) {
					int out_par = out_i*sizeV;

					float output = 0;
#pragma unroll			
					for (uint inp_i = 0; inp_i < sizeV; inp_i++)
					{		
						float param = const_area[out_par + inp_i];
						float val = inpVal[inp_i];
						output += param*val;
					}// inp_i

					//suppression filter

					//output = fmaxf(output, 0);
					vmax = fmaxf(output, vmax);
				}//out_i


				for (uint out_i = 0; out_i < sizeH; out_i++) {
					int out_par = out_i*sizeV;

					float output = 0;
#pragma unroll
					for (uint inp_i = 0; inp_i < sizeV; inp_i++)
					{		
						float param = const_area[out_par + inp_i];
						float val = inpVal[inp_i];
						output += param*val;
					}// inp_i

					//suppression filter
					output = fmaxf(output - SCALE_H*(vmax-output), 0);

					int toffset = color_off*sizeH + out_i*pix_tag_stride +  xy_off;
					target[toffset] = output;
				}//out_i

			}//color
        }
    }

}


template <int sizeV>
__global__ void kVectFuncGrad(const float* actGrad, const float* input, float* const target,
								const uint numPixelsPerGroup, const uint numCases,
								const uint strideInp, const uint strideOut,
								int numColors, int sizeH) {

	const int inStep = strideInp*numPixelsPerGroup;
	const int outStep = strideOut*numPixelsPerGroup;

	const int pix_out_stride = numPixelsPerGroup*strideOut;
	const int pix_in_stride = numPixelsPerGroup*strideInp;

	const int btx = blockDim.x*blockIdx.x + threadIdx.x;
	const int bty = blockDim.y*blockIdx.y + threadIdx.y;

	const int bd_off_in =  bty*strideInp + btx;
	const int bd_off_out = bty*strideOut + btx;

//with no N_SUM ix, iy == 0 almost always
    for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) {
        for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x) {	

			int xy_off_in = iy*strideInp +	ix + bd_off_in;
			int xy_off_out = iy*strideOut +	ix + bd_off_out;

			for (uint color = 0; color < numColors; color ++) {	//optimize away

				//Offset out_offset;
				//out_offset 
				//<< Index(color) << sizeH << numPixelsPerGroup << Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
				//<< strideOut
				//<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

				//Offset v_offset;
				//v_offset 
				//<< Index(color) << sizeV << numPixelsPerGroup << Index(iy) << Index(blockDim.y, blockIdx.y) << Index(threadIdx.y)
				//<< strideInp
				//<< Index(ix ) << Index(blockDim.x, blockIdx.x) << Index(threadIdx.x);

				float vres[sizeV];
				memset(vres, 0, sizeof(vres));

				for (uint out_i = 0; out_i < sizeH; out_i++)
				{
					int out_off = color*pix_out_stride*sizeH + out_i*pix_out_stride + xy_off_out;

					float vsum = 0;
					for (uint inp_i = 0; inp_i < sizeV; inp_i++) {	
					int in_off = color*pix_in_stride*sizeV + inp_i*pix_in_stride + xy_off_in;

						vsum += input[in_off]*const_area[out_i*sizeV + inp_i];
					}

					if(vsum > 0)
					{
						float grad_next = actGrad[out_off];

						for (uint inp_i = 0; inp_i < sizeV; inp_i++)
							vres[inp_i] += grad_next*const_area[out_i*sizeV + inp_i];
					}
				}

				for (uint inp_i = 0; inp_i < sizeV; inp_i++)
				{
					int in_off = color*pix_in_stride*sizeV + inp_i*pix_in_stride + xy_off_in;
					target[in_off] = vres[inp_i];
				}

			}//color
		}//ix
	}//iy
}

template <int sizeV>
__global__ void kVectFuncParamWeightGrad(	const float* actGrad, const float* input, float** const target,
											const uint numColors,
											const uint target_size, const uint numPixelsPerGroup, const uint numCases,
											const uint strideInp, const uint strideOut, const uint strideTag, int sizeH)
{
	extern __shared__ float sh_mem[];
	const int xy_off = threadIdx.y*blockDim.x + threadIdx.x;
	const int res_off = xy_off*sizeV*sizeH;
	float* resh = sh_mem + sizeV*blockDim.x*blockDim.y + res_off;
	float* in_store = sh_mem;
	
	memset(resh, 0, sizeV*sizeH*sizeof(float));

	const int btx = blockDim.x*blockIdx.x + threadIdx.x;
	const int bty = blockDim.y*blockIdx.y + threadIdx.y;

	const int bd_off_in =  bty*strideInp + btx;
	const int bd_off_out = bty*strideOut + btx;
	const int bd_off_tag = bty*strideTag + btx;

	const int pix_out_stride = numPixelsPerGroup*strideOut;
	const int pix_in_stride = numPixelsPerGroup*strideInp;
	
	for (uint iy = 0; iy < numPixelsPerGroup; iy += gridDim.y*blockDim.y) {

	  for (uint ix = 0; ix < numCases; ix += gridDim.x*blockDim.x) {	

		int xy_off_in = iy*strideInp +	ix + bd_off_in;
		int xy_off_out = iy*strideOut +	ix + bd_off_out;

		for (uint color = 0; color < numColors; color ++) {	

			float* inp_val = in_store + xy_off*sizeV;
			//float inp_val[sizeV];

				for (uint pin = 0; pin < sizeV; pin++)
				{
					int in_off = color*pix_in_stride*sizeV + pin*pix_in_stride + xy_off_in;
					inp_val[pin] = input[in_off];
				}

				int kmax= 0;
				float vmax = 0;

				for (uint pout = 0; pout < sizeH; pout++)
				{
					float vsum = 0;
#pragma unroll
					for (uint pin = 0; pin < sizeV; pin++)
					{
						vsum += inp_val[pin]*const_area[pout*sizeV + pin];
					}
					if(vsum > vmax)
					{
						vmax = vsum;
						kmax = pout;
					};
				}//pout

				float vres_max[sizeV];
				memset(vres_max, 0, sizeof(vres_max));

				for (uint pout = 0; pout < sizeH; pout++)
				{
					float* vres =  resh + sizeV*pout;

					int out_off = color*pix_out_stride*sizeH + pout*pix_out_stride + xy_off_out;
					float grad_next = actGrad[out_off];

					float output = 0;
#pragma unroll
					for (uint pin = 0; pin < sizeV; pin++)
					{
						output +=  inp_val[pin]*const_area[pout*sizeV + pin];
					}

					output = fmaxf(output - SCALE_H*(vmax-output), 0);

					if(output > 0)
					{
						for (uint pin = 0; pin < sizeV; pin++)
						{
							vres[pin] += grad_next*(1+SCALE_H)*inp_val[pin];
							vres_max[pin] += - SCALE_H*grad_next*inp_val[pin];
						}
					}//vsum
				}//pout
#pragma unroll
				for (uint pin = 0; pin < sizeV; pin++)
				{
					resh[kmax*sizeV + pin] += vres_max[pin];
				}

			}//color
		}//ix
	}//iy
		
	for (uint pout = 0; pout < sizeH; pout++)
#pragma unroll
	for (uint pin = 0; pin < sizeV; pin++)
	{

		target[pout*sizeV+pin][bd_off_tag] = resh[pout*sizeV+pin];
	}

}

//*************************************************************************************
//-------------------------------------------------------------
//API EltwiseMax
//-------------------------------------------------------------

void computeEltwiseMaxGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& output, NVMatrix& target, bool add) {
    assert(actGrad.isContiguous());
    assert(output.isContiguous());
    assert(input.isContiguous());
    assert(actGrad.isSameDims(input));
    assert(actGrad.isSameDims(output));
  
    dim3 blocks(DIVUP(actGrad.getNumElements(), 128));
    dim3 threads(128);
    if (add) {
        assert(actGrad.isSameDims(target));
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), true>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, true><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    } else {
        target.resize(actGrad);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), false>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, false><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    }
    
    cutilCheckMsg("computeEltwiseMaxGrad: Kernel execution failed");
}

//-------------------------------------------------------------
//API EltwiseFunc
//-------------------------------------------------------------



void computeEltwiseFuncAct(NVMatrix& input, NVMatrix& target, vector<double>& param, int size_in, int size_out)
{

	assert(size_in <= 4 || size_in == 6 || size_in == 8 || size_in == 12 || size_in == 16);
	//int height = input.getFollowingDim(), width = input.getLeadingDim();	
    //int numCases = input.getNumCols(); 
    //int numIn = input.getNumRows(); 

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

	int out_width = inp_width;
	int out_height = (inp_height*size_out)/size_in;

	//printf(" inp_height %i inp_width %i \n",inp_height, inp_width);
	//printf(" size_in %i size_out %i \n", size_in, size_out);
	//printf(" out_height %i out_width %i \n",out_height, out_width);

    if (target.getNumCols() != out_width || target.getNumRows() != out_height) {
        target.resize(out_height, out_width);
		//printf("**resize out_height %i out_width %i \n",out_height, out_width);
    }

	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);

	int numPixelsPerGroup = out_height/size_out;

    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);
    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(out_width, threads.x)),
                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

//debug
	//printf("kEltwiseFuncAct -------------\n");
	//printf("temp %f %f %f  %f %f %f \n", temp[0],temp[1],temp[2],temp[3],temp[4],temp[5]);
	//input.nan2zero();
	//float sum = input.sum();
	//printf(" size_in %i size_out %i sum %f \n", size_in, size_out, sum);
//	const int numPixelsPerGroup1 = inp_height/size_in;
//	printf(" numPixelsPerGroup %i numPixelsPerGroup1 %i target.getNumRows %i \n", numPixelsPerGroup, numPixelsPerGroup1, target.getNumRows());
//	//hipMemset(target.getDevData(), 0, target.getNumElements()*sizeof(float));
//printf(" target.getStride() %i target.getNumRows() %i target.getNumCols() %i \n", target.getStride(), target.getNumRows(), target.getNumCols());

	//kEltwiseFuncAct_t<ELTWISE_THREADS_X, ELTWISE_THREADS_Y, 3><<<blocks, threads>>>(input.getDevData(),
	//target.getDevData(), inp_height, inp_width, input.getStride(), target.getStride(), size_in, size_out);
	//float sumt0 = target.sum();
	//printf("kEltwiseFuncAct_t sumt_0 %f \n", sumt0);


#define ELT_ACT(SIZE_ARR) \
	if(size_in == SIZE_ARR){\
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseFuncAct<SIZE_ARR>), hipFuncCachePreferL1);\
	kEltwiseFuncAct<SIZE_ARR><<<blocks, threads>>>(input.getDevData(),\
	target.getDevData(), inp_height, inp_width, input.getStride(), target.getStride(), size_in, size_out);};
	ELT_ACT(1)
	ELT_ACT(2)
	ELT_ACT(3)
	ELT_ACT(4)
	ELT_ACT(6)
	ELT_ACT(8)
	ELT_ACT(12)
	ELT_ACT(16)
#undef ELT_ACT

//float sumt = target.sum();
//	printf("kEltwiseFuncAct sumt %f \n", sumt);

	cutilCheckMsg("computeEltwiseFuncAct: Kernel execution failed");
}

void computeEltwiseFuncGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& target,
								 vector<double>& param, int size_in, int size_out)
{


	assert(size_out <= 4 || size_out == 6 || size_out == 8 || size_out == 12 || size_out == 16);
	//int height = input.getFollowingDim(), width = input.getLeadingDim();	
    //int numCases = input.getNumCols(); 
    //int numIn = input.getNumRows(); 

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

    if (target.getNumCols() != inp_width || target.getNumRows() != inp_height) {
        target.resize(inp_height, inp_width);
    }

	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);


	int numPixelsPerGroup = inp_height/size_in;

    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);

    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(inp_width, threads.x)),
                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

	//printf("computeEltwiseFuncGrad numPixelsPerGroup %i --------------------\n", numPixelsPerGroup);
	//float sumA = actGrad.sum();
	//float sumI = input.sum();

	//printf("sum actGrad %f input %f \n", sumA, sumI);
	//printf(" size_in %i size_out %i tag size %i sumt %f \n", size_in, size_out,  target.getNumElements());
	//printf(" target.getStride() %i actGrad  %i input %i \n", target.getStride(), actGrad.getNumRows(), input.getNumRows());

	//kEltwiseFuncGrad_t<ELTWISE_THREADS_X, ELTWISE_THREADS_Y, 3><<<blocks, threads>>>(actGrad.getDevData(),
	//	input.getDevData(), target.getDevData(), inp_height, inp_width,
	//	input.getStride(), actGrad.getStride(), size_in, size_out);

	//float sumtt = target.sum();
	//printf("sum_test_tag %f \n", sumtt);


#define ELT_GRAD(SIZE_ARR) \
		if(size_out == SIZE_ARR){\
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseFuncGrad<SIZE_ARR>), hipFuncCachePreferL1);\
			kEltwiseFuncGrad<SIZE_ARR><<<blocks, threads>>>(actGrad.getDevData(),\
				input.getDevData(), target.getDevData(), inp_height, inp_width,\
				input.getStride(), actGrad.getStride(), size_in, size_out);};
		ELT_GRAD(1)
		ELT_GRAD(2)
		ELT_GRAD(3)
		ELT_GRAD(4)
		ELT_GRAD(6)
		ELT_GRAD(8)
		ELT_GRAD(12)
		ELT_GRAD(16)
#undef ELT_GRAD

//	float sumt = target.sum();
//	printf("sum_tag %f \n", sumt);


	cutilCheckMsg("computeEltwiseFuncGrad: Kernel execution failed");
};

void computeEltwiseFuncParamGradSingle(NVMatrix& actGrad, NVMatrix& input,
								 NVMatrix& target, NVMatrix& target_m,
								 int pin, int pout, int size_in, int size_out)
{

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();


	int numPixelsPerGroup = inp_height/size_in;
//	printf("inp_height %i numPixelsPerGroup %i \n", inp_height, numPixelsPerGroup);
#define N_SUM 1
    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);
    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(inp_width, threads.x)),
                std::min(NUM_BLOCKS_MAX, (int)DIVUP(numPixelsPerGroup/N_SUM, ELTWISE_THREADS_Y)));
#undef N_SUM

	int sizeX = blocks.x*threads.x;
	int sizeY = blocks.y*threads.y;

    if (target.getNumCols() != sizeX || target.getNumRows() != sizeY) {
		//printf(" tresize %i %i \n", sizeX, sizeY);
        target.resize(sizeY, sizeX);// numRows, numCols !
    }

	//printf(" target.getNumCols() %i target.getNumRows() %i elem %i   \n", target.getNumCols(), target.getNumRows(), target.getNumElements());

	//printf(" aft memset\n");
	//float rr = target.sum();
	//printf(" sum %f \n", rr);

    if (!target_m.isSameDims(target)) {
        target_m.resize(target);
    }
	//hipMemset(target_m.getDevData(), 0, sizeX*sizeY*sizeof(float));
	//hipMemset(target.getDevData(), 0, sizeX*sizeY*sizeof(float));	

	//printf(" target.getStride() %i sizeX %i sizeY %i target.isTrans() %i actGrad.getStride() %i \n", 
	//	target.getStride(),sizeX, sizeY, target.isTrans(), actGrad.getStride());


	//printf(" numPixelsPerGroup %i actGrad.getNumRows %i \n", numPixelsPerGroup, actGrad.getNumRows());

	//float ar1 = actGrad.sum();
	//float ir2 = input.sum();
	//printf("sum actGrad  %f input %f \n", ar1, ir2);


	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseFuncParamGradSingle), hipFuncCachePreferL1);

	//kEltwiseFuncParamGradSingle_t<ELTWISE_THREADS_X, ELTWISE_THREADS_Y><<<blocks, threads>>>(actGrad.getDevData(),
	//	input.getDevData(), target.getDevData(), target_m.getDevData(),
	//	pin, pout, inp_height, inp_width,
	//	input.getStride(), actGrad.getStride(), target.getStride(),
	//	size_in, size_out);

	//float rr11 = target.sum();
	//float rr21 = target_m.sum();
	//printf(" sum aft %f %f \n", rr11, rr21);
	//printf("sum1 actGrad  %f input %f \n", ar1, ir2);

	kEltwiseFuncParamGradSingle<<<blocks, threads>>>(actGrad.getDevData(),
		input.getDevData(), target.getDevData(), target_m.getDevData(),
		pin, pout, inp_height, inp_width,
		input.getStride(), actGrad.getStride(), target.getStride(),
		size_in, size_out);

	//float rr1 = target.sum();
	//float rr2 = target_m.sum();
	//printf(" sum1 aft %f %f \n", rr1, rr2);

 /*       int height = input0.getFollowingDim(), width = input0.getLeadingDim();

        dim3 blocks(std::min(NUM_BLOCKS_MAX, DIVUP(width, ELTWISE_THREADS_X)),
                    std::min(NUM_BLOCKS_MAX, DIVUP(height, ELTWISE_THREADS_Y)));
        dim3 threads(ELTWISE_THREADS_X, ELTWISE_THREADS_Y);

		int sizeX = blocks.x*threads.x;
		int sizeY = blocks.y*threads.y;

        if (target0.getNumRows() != sizeX || target0.getNumCols() !=  sizeY) {
            target0.resize(sizeX, sizeY);
        }

		////debug
  //      if (!target0.isSameDims(input0)) {
  //          target0.resize(input0);
  //      }//shortening is not working

        if (!target1.isSameDims(target0)) {
            target1.resize(target0);
        }

        if (!target2.isSameDims(target0)) {
            target2.resize(target0);
        }

        if (!target3.isSameDims(target0)) {
            target3.resize(target0);
        }

        if (!target4.isSameDims(target0)) {
            target4.resize(target0);
        }

        if (!target5.isSameDims(target0)) {
            target5.resize(target0);
        }

		hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseFuncParamGrad<ELTWISE_THREADS_X), ELTWISE_THREADS_Y>, hipFuncCachePreferL1);

		kEltwiseFuncParamGrad<ELTWISE_THREADS_X, ELTWISE_THREADS_Y><<<blocks, threads>>>(actGrad.getDevData(), 
			input0.getDevData(), input1.getDevData(), input2.getDevData(),
			target0.getDevData(), target1.getDevData(), target2.getDevData(), target3.getDevData(), target4.getDevData(), target5.getDevData(),
			height, width, input0.getStride(), sizeX);
*/
		cutilCheckMsg("kEltwiseFuncParamGrad: Kernel execution failed");
};
//-------------------------------------------------------------
//API MicroConv
//-------------------------------------------------------------

#include "conv_debug.h"
#define SIZE_CONV 3


void computeMicroConvAct(NVMatrix& input, NVMatrix& target, vector<double>& param, int sizeModuleSide, int channels,
						 int imgSize, int imgPixels, int numFilters)
{
	int out_width = input.getNumCols();
	int out_height = input.getNumRows()*numFilters;

    if (target.getNumCols() != out_width || target.getNumRows() != out_height) {
        target.resize(out_height, out_width);
		//printf("**resize out_height %i out_width %i \n",out_height, out_width);
    }

	int numCases = out_width;

	int imgSizeX = imgSize;
	int imgSizeY = imgSize;

	int img_threads_x = 8;
	int img_threads_y = 8;
	int casePerThread = 16;
	int nblocksx = 2;//~number of blocks x
	int case_threads = DIVUP(numCases, nblocksx*casePerThread); 

	int imgBlocksY = DIVUP(imgSizeY,img_threads_x);
	int imgBlocksX = DIVUP(imgSizeX,img_threads_y);

	int lobe = sizeModuleSide/2;


	int sharedX = lobe*2 + img_threads_x;
	int sharedY = lobe*2 + img_threads_y;
	int shared_size = sharedX*sharedY*channels*case_threads*sizeof(float);

	dim3 threads(case_threads, img_threads_x*img_threads_y);
	dim3 blocks = dim3(DIVUP(numCases, threads.x*casePerThread), imgBlocksY*imgBlocksX);


	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);

	//printf("blocks.x %i blocks.y %i threads.x %i threads.y %i shared_size %i casePerThread %i\n",
	//	blocks.x, blocks.y, threads.x, threads.y, shared_size, casePerThread);
	//printf("sharedY %i img_threads_x %i img_threads_y %i sizeModuleSide %i imgSizeX %i imgSizeY %i imgPixels %i numFilters %i numCases %i lobe %i\n",
	//	sharedY,img_threads_x,img_threads_y,sizeModuleSide,imgSizeX,imgSizeY, imgPixels,numFilters,numCases,lobe);


	assert(SIZE_CONV == 3);

	//singletonTempMem.allocFloatElement(input.getNumCols()*input.getNumRows());
	//singletonTempMem.allocFloatElement(out_height*out_width);
	//float* tempHostInput = singletonTempMem.getPtr(0);
	//float* tempHostTarget = singletonTempMem.getPtr(1);
	//int deltan = singletonTempMem._start[1]-singletonTempMem._start[0];
	//printf(" size inp %i singletonTempMem._size %i deltan %i \n",
	//	input.getNumCols()*input.getNumRows(),singletonTempMem._size, deltan);
	//cutilSafeCallNoSync( hipMemcpy(tempHostInput, input.getDevData(), input.getNumCols()*input.getNumRows()*sizeof(float), hipMemcpyDeviceToHost) );
	//double sum_host =0;
	//debugMicroConvFilterAct((SIZE_CONV-1)/2, SIZE_CONV, temp, tempHostInput, tempHostTarget,
	//									numCases, channels, numFilters,
	//									sharedY, img_threads_x,  img_threads_y, 
	//									imgSizeX, imgSizeY,
	//									imgPixels);
	// sum_host = Sum(tempHostTarget, out_height*out_width);
	//printf(" debugMicroConvFilterAct sum %f \n", sum_host);


	//emuMicroConvFilterAct(threads.x, threads.y, blocks.x, blocks.y,
	//									(SIZE_CONV-1)/2, SIZE_CONV,
	//									temp, tempHostInput, tempHostTarget,
	//									numCases, channels, numFilters, casePerThread,
	//									sharedY, img_threads_x,  img_threads_y, 
	//									imgSizeX, imgSizeY,
	//									imgPixels);

	//sum_host = Sum(tempHostTarget, out_height*out_width);
	//printf(" emuMicroConvFilterAct sum %f \n", sum_host);


	//singletonTempMem.reset();



	kMicroConvFilterAct<(SIZE_CONV-1)/2, SIZE_CONV><<<blocks, threads, shared_size>>>(input.getDevData(), target.getDevData(),
										numCases, channels, numFilters, casePerThread,
										sharedY, img_threads_x,  img_threads_y, 
										imgSizeX, imgSizeY,
										imgPixels);

//debug
	//printf("kMicroConvAct4Channel end \n");

	//float sum = target.sum();
	//printf(" kMicroConvAct4Channel sum %f \n", sum);

	cutilCheckMsg("computeMicroConvAct: Kernel execution failed");

};

void computeMicroConvActGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& target,
							 vector<double>& param, int sizeModuleSide, int channels,
							int imgSize, int imgPixels, int numFilters)
{


    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

    if (target.getNumCols() != inp_width || target.getNumRows() != inp_height) {
        target.resize(inp_height, inp_width);
    }

	int numCases = inp_width;

	int imgSizeX = imgSize;
	int imgSizeY = imgSize;

	int img_threads_x = 8;
	int img_threads_y = 8;
	int casePerThread = 16;

	int nblocksx = 2;//~number of blocks x

	int case_threads = DIVUP(numCases, nblocksx*casePerThread); 

	int lobe = sizeModuleSide/2;

	int sharedX = lobe*2 + img_threads_x;
	int sharedY = lobe*2 + img_threads_y;
	int shared_size = sharedX*sharedY*numFilters*channels*case_threads*sizeof(float);

	int imgBlocksY = DIVUP(imgSizeY,img_threads_x);
	int imgBlocksX = DIVUP(imgSizeX,img_threads_y);

	dim3 threads(case_threads, img_threads_x*img_threads_y);
	dim3 blocks = dim3(DIVUP(numCases, threads.x*casePerThread), imgBlocksY*imgBlocksX);
	

	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);


	printf("blocks.x %i blocks.y %i threads.x %i threads.y %i shared_size %i casePerThread %i\n",
		blocks.x, blocks.y, threads.x, threads.y, shared_size, casePerThread);
	printf("sharedY %i img_threads_x %i img_threads_y %i sizeModuleSide %i imgSizeX %i imgSizeY %i imgPixels %i numFilters %i numCases %i lobe %i\n",
		sharedY,img_threads_x,img_threads_y,sizeModuleSide,imgSizeX,imgSizeY, imgPixels,numFilters,numCases,lobe);


	//singletonTempMem.allocFloatElement(actGrad.getNumCols()*actGrad.getNumRows());
	//singletonTempMem.allocFloatElement(target.getNumCols()*target.getNumRows());
	//float* tempHostInput = singletonTempMem.getPtr(0);
	//float* tempHostTarget = singletonTempMem.getPtr(1);

	//cutilSafeCallNoSync( hipMemcpy(tempHostInput, actGrad.getDevData(), actGrad.getNumCols()*actGrad.getNumRows()*sizeof(float),
	//	hipMemcpyDeviceToHost) );

	//double sum_host =0;
	//debugMicroConvActGrad((SIZE_CONV-1)/2, SIZE_CONV, temp, tempHostInput, tempHostTarget,
	//							numCases, channels, numFilters, casePerThread, 
	//							img_threads_x, img_threads_y,
	//							sharedY, sizeModuleSide, lobe,
	//							imgSizeX, imgSizeY,
	//							imgPixels);
	//sum_host = Sum(tempHostTarget, target.getNumCols()*target.getNumRows());
	//printf(" debugMicroConvFilterAct sum %f \n", sum_host);
	//singletonTempMem.reset();

	kMicroConvActGrad<<<blocks, threads, shared_size>>>(actGrad.getDevData(), target.getDevData(),
								numCases, channels, numFilters, casePerThread, 
								img_threads_x, img_threads_y,
								sharedY, sizeModuleSide, lobe,
								imgSizeX, imgSizeY,
								imgPixels);
//	double sum = target.sum();
//	printf(" kMicroConvGrad sum %f \n", sum);
//	printf("kMicroConvGrad end \n");

	cutilCheckMsg("kMicroConvGrad: Kernel execution failed");
}

void computeMicroConvWeightGrad(NVMatrix& actGrad, NVMatrix& input,
								vector<NVMatrix>& tempMatrix, void* arrayPtr,
								vector<double>& param, int sizeModuleSide, int channels,
								int imgSize, int imgPixels, int numFilters)
{

	int numCases = input.getNumCols();

	int imgSizeX = imgSize;
	int imgSizeY = imgSize;

	int img_threads_x = 8;
	int img_threads_y = 8;
	int casePerThread = 16;

	int nblocksx = 2;//~number of blocks x

	int case_threads = DIVUP(numCases, nblocksx*casePerThread); 

	int lobe = sizeModuleSide/2;

	int sharedX = lobe*2 + img_threads_x;
	int sharedY = lobe*2 + img_threads_y;

	int conv_size = (lobe*2 + 1);
	int conv_size2 = conv_size*conv_size;

	int imgBlocksY = DIVUP(imgSizeY,img_threads_x);
	int imgBlocksX = DIVUP(imgSizeX,img_threads_y);

//for optimization can change both block sizes!
	dim3 threads(case_threads, img_threads_x*img_threads_y);
	dim3 blocks = dim3(DIVUP(numCases, threads.x*casePerThread), imgBlocksY*imgBlocksX);

	int sizeSharedBlock = sharedX*sharedY;
	int shared_size = (sizeSharedBlock*threads.x + threads.x*threads.y*numFilters*conv_size2)*sizeof(float);//looped out - case_threads*imgsPerThread;

    int tag_width = DIVUP(input.getNumCols(), casePerThread) ; //could be reduced
    int tag_height = blocks.y*threads.y;//could be reduced
	int tag_size = tag_width*tag_height;

	float* tempMatrixPtr[CONST_AREA_SIZE];
	for(int i =0; i < tempMatrix.size(); i++)
	{
		if (tempMatrix[i].getNumCols() != tag_width || tempMatrix[i].getNumRows() != tag_height) {
			tempMatrix[i].resize(tag_height, tag_width);
			hipMemset(tempMatrix[i].getDevData(), 0, tag_size*sizeof(float));
		}

		tempMatrixPtr[i] = tempMatrix[i].getDevData();
	}

	hipMemcpy(arrayPtr, tempMatrixPtr, sizeof(float*)*tempMatrix.size(), hipMemcpyHostToDevice);

//	printf("kMicroConvWeightGrad *************** \n");
//	printf("tag_width %i tag_height %i shared_size %i  tempMatrix.size() %i conv_size %i casePerThread %i\n",
//		tag_width, tag_height, shared_size, tempMatrix.size(), conv_size, casePerThread);
//
//	printf("blocks.x %i blocks.y %i threads.x %i threads.y %i shared_size %i \n",
//		blocks.x, blocks.y, threads.x, threads.y, shared_size);
//	printf("sharedY %i img_threads_x %i img_threads_y %i sizeModuleSide %i imgSizeX %i imgSizeY %i imgPixels %i numFilters %i numCases %i lobe %i\n",
//		sharedY,img_threads_x,img_threads_y,sizeModuleSide,imgSizeX,imgSizeY, imgPixels,numFilters,numCases,lobe);
//
//
	//const int sizeConv2 = SIZE_CONV*SIZE_CONV;
	//int filterID = 0;
	//int dsy = 0;
	//int dsx = 1;
	//int channelID = 0;
	//int ind_coeff = filterID*sizeConv2 + (dsy + lobe)*SIZE_CONV +(dsx + lobe);
//
//	singletonTempMem.allocFloatElement(actGrad.getNumCols()*actGrad.getNumRows());
//	singletonTempMem.allocFloatElement(input.getNumCols()*input.getNumRows());
//	singletonTempMem.allocFloatElement(tag_height*tag_width);
//	int out_width = input.getNumCols();
//	int out_height = input.getNumRows()*numFilters;
//	singletonTempMem.allocFloatElement(out_width*out_height);
//
//	float* tempHostAct = singletonTempMem.getPtr(0);
//	float* tempHostInp = singletonTempMem.getPtr(1);
//	float* tempHostTag = singletonTempMem.getPtr(2);
//	float* tempHostTagA = singletonTempMem.getPtr(3);
//
//	hipMemcpy(tempHostAct, actGrad.getDevData(), actGrad.getNumCols()*actGrad.getNumRows()*sizeof(float),
//		hipMemcpyDeviceToHost);
//
//	hipMemcpy(tempHostInp, input.getDevData(), input.getNumCols()*input.getNumRows()*sizeof(float),
//		hipMemcpyDeviceToHost);
//	//memset(tempHostTagA, 0, tag_height*tag_width*sizeof(float));
//	memset(tempHostTag, 0, tag_height*tag_width*sizeof(float));
//
//	double sum_a = Sum(tempHostAct, actGrad.getNumCols()*actGrad.getNumRows());
//	double sum_i = Sum(tempHostInp, input.getNumCols()*input.getNumRows());
//	 printf(" sum_a %f sum_i %f \n", sum_a, sum_i);
//
//	float temp[CONST_AREA_SIZE];
//	assert(param.size() <= CONST_AREA_SIZE);
//	memset(temp, 0, sizeof(temp));
//	for(int i = 0; i < param.size(); i++)
//		temp[i] = (float)param[i];
//
//	
//	debugMicroConvLinApprox((SIZE_CONV-1)/2, SIZE_CONV, temp, tempHostInp, tempHostAct, tempHostTagA,
//										numCases, channels, numFilters,
//										sharedY, img_threads_x,  img_threads_y, 
//										imgSizeX, imgSizeY,
//										imgPixels);
//	double sum_host0 = Sum(tempHostTagA, out_height*out_width);
//	printf(" debugMicroConvFilterAct sum0 %f \n", sum_host0);
//	double delta = 1e-3;
//	temp[ind_coeff] += delta;
//
//	debugMicroConvLinApprox((SIZE_CONV-1)/2, SIZE_CONV, temp, tempHostInp, tempHostAct, tempHostTagA,
//										numCases, channels, numFilters,
//										sharedY, img_threads_x,  img_threads_y, 
//										imgSizeX, imgSizeY,
//										imgPixels);
//	double sum_host1 = Sum(tempHostTagA, out_height*out_width);
//	printf(" debugMicroConvFilterAct sum1 %f \n", sum_host1);
//
//	printf(" debugMicroConv grad %f \n", (sum_host1-sum_host0)/delta);
//
//
//memset(tempHostTag, 0, tag_height*tag_width*sizeof(float));
//  debugMicroConvWeightGrad(lobe, SIZE_CONV, dsx, dsy, filterID, channelID, tempHostAct, tempHostInp, tempHostTag,
//								tag_size, numCases,
//								channels, numFilters, 
//								img_threads_x, img_threads_y, sharedY,
//								lobe, sizeModuleSide, sizeSharedBlock,
//								imgSizeX, imgSizeY, imgPixels);
//
//  double sum_host = Sum(tempHostTag, tag_height*tag_width);
//  printf(" debugMicroConvWeightGrad sum %f \n", sum_host);
//
//memset(tempHostTag, 0, tag_height*tag_width*sizeof(float));

  //emuMicroConvWeightGrad(threads.x, threads.y, blocks.x, blocks.y,
		//					lobe, SIZE_CONV, dsx, dsy, filterID, channelID, tempHostAct, tempHostInp, tempHostTag,
		//						tag_size, numCases, casePerThread, tag_width,
		//						channels, numFilters, 
		//						img_threads_x, img_threads_y, sharedY,
		//						sizeSharedBlock,
		//						imgSizeX, imgSizeY, imgPixels);

  // double sum_host_emu = Sum(tempHostTag, tag_height*tag_width);
  //printf(" emuMicroConvWeightGrad sum %f \n", sum_host_emu);


	kMicroConvWeightGrad<SIZE_CONV/2><<<blocks, threads, shared_size>>>(actGrad.getDevData(), input.getDevData(), (float**)arrayPtr,
								tag_size, numCases, casePerThread, tag_width,
								channels, numFilters, 
								img_threads_x, img_threads_y,
								imgSizeX, imgSizeY, imgPixels);

//	double sum_ag = actGrad.sum();
//	double sum_ig = input.sum();
//double sum = tempMatrix[ind_coeff].sum();
//printf(" kMicroConvWeightGrad sum %f  \n", sum);
	//printf(" kMicroConvWeightGrad sum %f sum_ag %f sum_ig %f \n", sum, sum_ag, sum_ig);

////debug
//	printf("kMicroConvWeightGrad end \n");

	cutilCheckMsg("kMicroConvWeightGrad: Kernel execution failed");
}

//-------------------------------------------------------------
//API VectFunc
//-------------------------------------------------------------

void computeVectFuncAct(NVMatrix& input, NVMatrix& target, vector<double>& param, int sizeV, int sizeH, int channels)
{
//printf("\n kVectFuncAct start*** \n");

	assert(sizeV <= 4 || sizeV == 6 || sizeV == 8 || sizeV == 12 || sizeV == 16);

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

	int out_width = inp_width;
	int out_height = (inp_height*sizeH)/sizeV;

	int numCases = out_width;
	int numPixelsPerGroup = inp_height/channels;

	int numColors = channels/sizeV;


    if (target.getNumCols() != out_width || target.getNumRows() != out_height) {
//		printf("**resize out_height %i out_width %i \n",out_height, out_width);
        target.resize(out_height, out_width);
    }
	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);

    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);

    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(inp_width, threads.x)),
                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

	for(int i = 0; i < param.size()/2; i++)
	{
		printf("param %f %f \n",  param[2*i], param[2*i]);
	}

//	float sumi = input.sum();
//	printf("sumi %f \n",  sumi);
//	printf("blocks.x %i blocks.y %i threads.x %i threads.y %i numColors %i \n",blocks.x, blocks.y, threads.x, threads.y, numColors);
//	printf("inp_height %i numPixelsPerGroup %i out_width %i out_height %i sizeV %i \n",inp_height, numPixelsPerGroup,out_width,out_height,sizeV);
//	printf("sizeV %i sizeH %i strides %i %i \n", sizeV, sizeH, input.getStride(), target.getStride());
////debug
//	hipMemset(target.getDevData(), 0, out_height*out_width*sizeof(float));
//	
//	singletonTempMem.allocFloatElement(input.getNumCols()*input.getNumRows());
//	singletonTempMem.allocFloatElement(out_height*out_width);
//	float* tempHostInput = singletonTempMem.getPtr(0);
//	float* tempHostTarget = singletonTempMem.getPtr(1);
//	hipMemcpy(tempHostInput, input.getDevData(), input.getNumCols()*input.getNumRows()*sizeof(float), hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();
//
//	double sum_inp = Sum(tempHostInput, input.getNumCols()*input.getNumRows());
//	printf("sum_inp %f \n",  sum_inp);
//
//	double sum_host =0;
//	memset(tempHostTarget, 0, out_height*out_width*sizeof(float));
//	debugVectFuncAct(sizeV, temp, tempHostInput, tempHostTarget,
//								numPixelsPerGroup, numCases, input.getStride(), target.getStride(), numColors, sizeH);
//
//	sum_host = Sum(tempHostTarget, out_height*out_width);
//
//	printf(" debugVectFuncAct sum %f \n", sum_host);
//
//	singletonTempMem.reset();

#define ELT_ACT(SIZE_ARR) \
	if(sizeV == SIZE_ARR){\
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kVectFuncAct<SIZE_ARR>), hipFuncCachePreferL1);\
	kVectFuncAct<SIZE_ARR><<<blocks, threads>>>(input.getDevData(),\
	target.getDevData(), numPixelsPerGroup, numCases, input.getStride(), target.getStride(), numColors, sizeH);};
	ELT_ACT(1)
	ELT_ACT(2)
	ELT_ACT(3)
	ELT_ACT(4)
	ELT_ACT(6)
	ELT_ACT(8)
	ELT_ACT(12)
	ELT_ACT(16)
#undef ELT_ACT

//	float sumt = target.sum();
//	printf("kVectFuncAct sumt %f \n",  sumt);

	//printf("kVectFuncAct end \n");
	cutilCheckMsg("kVectFuncAct: Kernel execution failed");
}


void computeVectFuncGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& target,
								 vector<double>& param,  int sizeV, int sizeH, int channels)
{


	assert(sizeV <= 4 || sizeV == 6 || sizeV == 8 || sizeV == 12 || sizeV == 16);

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

    if (target.getNumCols() != inp_width || target.getNumRows() != inp_height) {
        target.resize(inp_height, inp_width);
    }


	int out_width = inp_width;
	int out_height = (inp_height*sizeH)/sizeV;

	int numCases = out_width;
	int numPixelsPerGroup = inp_height/channels;

	int numColors = channels/sizeV;

	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);

    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);

    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(inp_width, threads.x)),
                std::min(NUM_BLOCKS_MAX, DIVUP(numPixelsPerGroup, ELTWISE_THREADS_Y)));

	printf("kVectFuncGrad start ************************\n");
	printf("blocks.x %i blocks.y %i threads.x %i threads.y %i \n",
		blocks.x, blocks.y, threads.x, threads.y);
	printf("numPixelsPerGroup %i numCases %i numColors %i out_width %i out_height %i\n",
		numPixelsPerGroup, numCases, numColors, out_width, out_height);

	singletonTempMem.allocFloatElement(input.getNumCols()*input.getNumRows());
	singletonTempMem.allocFloatElement(inp_height*inp_width);
	singletonTempMem.allocFloatElement(actGrad.getNumCols()*actGrad.getNumRows());
	singletonTempMem.allocFloatElement(inp_height*inp_width);
	float* tempHostInput = singletonTempMem.getPtr(0);
	float* tempHostTarget = singletonTempMem.getPtr(1);
	float* tempHostActGrad = singletonTempMem.getPtr(2);
	float* tempHostTarget1 = singletonTempMem.getPtr(1);
	hipMemcpy(tempHostInput, input.getDevData(), input.getNumCols()*input.getNumRows()*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(tempHostActGrad, actGrad.getDevData(), actGrad.getNumCols()*actGrad.getNumRows()*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	debugVectFuncGrad(sizeV, temp, tempHostActGrad,
				tempHostInput, tempHostTarget, tempHostTarget1, numPixelsPerGroup, numCases,
				input.getStride(), actGrad.getStride(), numColors, sizeH);

	double sum_host = Sum(tempHostTarget, inp_height*inp_width);
	double sum_host1 = Sum(tempHostTarget1, inp_height*inp_width);
	printf(" debugVectFuncAct sum %f sum1 %f \n", sum_host, sum_host1);
	singletonTempMem.reset();

#define ELT_GRAD(SIZE_ARR) \
		if(sizeV == SIZE_ARR){\
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kVectFuncGrad<SIZE_ARR>), hipFuncCachePreferL1);\
			kVectFuncGrad<SIZE_ARR><<<blocks, threads>>>(actGrad.getDevData(),\
				input.getDevData(), target.getDevData(), numPixelsPerGroup, numCases,\
				input.getStride(), actGrad.getStride(), numColors, sizeH);};
		ELT_GRAD(1)
		ELT_GRAD(2)
		ELT_GRAD(3)
		ELT_GRAD(4)
		ELT_GRAD(6)
		ELT_GRAD(8)
		ELT_GRAD(12)
		ELT_GRAD(16)
#undef ELT_GRAD

	float sumt = target.sum();
	printf("kVectFuncGrad sum_tag %f \n", sumt);


	cutilCheckMsg("kVectFuncGrad: Kernel execution failed");

};

void computeVectFuncWeightGrad(NVMatrix& actGrad, NVMatrix& input,
								vector<NVMatrix>& tempMatrix,
								void* arrayPtr,
								vector<double>& param,  int sizeV, int sizeH, int channels)
{
	assert(sizeV <= 4 || sizeV == 6 || sizeV == 8 || sizeV == 12 || sizeV == 16);

    int inp_width = input.getNumCols(); 
    int inp_height = input.getNumRows();

	int out_width = inp_width;
	int out_height = (inp_height*sizeH)/sizeV;

	int numCases = out_width;
	int numPixelsPerGroup = inp_height/channels;

	int numColors = channels/sizeV;



#define N_SUM 1
    dim3 threads(min(ELTWISE_THREADS_X, inp_width), ELTWISE_THREADS_Y);
    dim3 blocks(std::min(NUM_BLOCKS_MAX, (int)DIVUP(inp_width, threads.x)),//reduce
                std::min(NUM_BLOCKS_MAX, (int)DIVUP(numPixelsPerGroup/N_SUM, ELTWISE_THREADS_Y)));
#undef N_SUM

	int shared_size = sizeV*(sizeH+1)*threads.x*threads.y*sizeof(float);

    int tag_width = blocks.x*threads.x; //could be reduced
    int tag_height = blocks.y*threads.y;//could be reduced
	int tag_size = tag_width*tag_height;

	float temp[CONST_AREA_SIZE];
	assert(param.size() <= CONST_AREA_SIZE);
	memset(temp, 0, sizeof(temp));
	for(int i = 0; i < param.size(); i++)
		temp[i] = (float)param[i];
	hipMemcpyToSymbol(HIP_SYMBOL(const_area), temp, sizeof(float)*CONST_AREA_SIZE, 0, hipMemcpyHostToDevice);

	float* tempMatrixPtr[CONST_AREA_SIZE];
	for(int i =0; i < tempMatrix.size(); i++)
	{
		if (tempMatrix[i].getNumCols() != tag_width || tempMatrix[i].getNumRows() != tag_height) {
			tempMatrix[i].resize(tag_height, tag_width);
		}
		tempMatrixPtr[i] = tempMatrix[i].getDevData();
	}

	hipMemcpy(arrayPtr, tempMatrixPtr, sizeof(float*)*tempMatrix.size(), hipMemcpyHostToDevice);

	//for(int i =0; i < tempMatrix.size(); i++)
	//{
	//	hipMemset(tempMatrix[i].getDevData(), 0, tag_size*sizeof(float));
	//}
//----------
	//printf("kVectFuncParamWeightGrad start ************************\n");

	//printf("blocks.x %i blocks.y %i threads.x %i threads.y %i shared_size %i \n",
	//	blocks.x, blocks.y, threads.x, threads.y, shared_size);
	//printf("numPixelsPerGroup %i numCases %i numColors %i out_width %i out_height %i\n",
	//	numPixelsPerGroup, numCases, numColors, out_width, out_height);

	////float sumi = input.sum();
	////printf("sumi %f \n",  sumi);

	//printf( "tempMatrix.size() %i tag_width %i tag_height %i actGrad %i %i tempMatrix[0].getStride() %i \n",
	//		tempMatrix.size(), tag_width, tag_height, actGrad.getNumCols(), actGrad.getNumRows(), tempMatrix[0].getStride());
	//
	//singletonTempMem.allocFloatElement(input.getNumCols()*input.getNumRows());
	//singletonTempMem.allocFloatElement(max(tag_height*tag_width, out_height*out_width));
	//singletonTempMem.allocFloatElement(actGrad.getNumCols()*actGrad.getNumRows());
	//float* tempHostInput = singletonTempMem.getPtr(0);
	//float* tempHostTarget = singletonTempMem.getPtr(1);
	//float* tempHostActGrad = singletonTempMem.getPtr(2);
	//hipMemcpy(tempHostInput, input.getDevData(), input.getNumCols()*input.getNumRows()*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(tempHostActGrad, actGrad.getDevData(), actGrad.getNumCols()*actGrad.getNumRows()*sizeof(float), hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();


	//debugVectFuncParamWeightGrad(sizeV,  temp, blocks.y, threads.y, blocks.x, threads.x, 
	//			tempHostActGrad,  tempHostInput, tempHostTarget, numColors, tag_size, numPixelsPerGroup, numCases,
	//			input.getStride(), actGrad.getStride(), tempMatrix[0].getStride(), sizeH);


	//double sum_host = Sum(tempHostTarget, tag_height*tag_width);
	//	double sum_act = Sum(tempHostActGrad, actGrad.getNumCols()*actGrad.getNumRows());
	//singletonTempMem.reset();

	//float suma = actGrad.sum();
	//printf("debugVectFuncParamWeightGrad******* sum_host %f sum_act %f suma %f\n", sum_host, sum_act, suma);


	//debugVectFuncLinApprox(sizeV, temp, tempHostInput,
	//							tempHostActGrad, tempHostTarget,
	//							numPixelsPerGroup, numCases,
	//							input.getStride(), tempMatrix[0].getStride(), numColors, sizeH);
	//float delta = 1e-4;
	//float sumLA0 =  Sum(tempHostTarget, out_height*out_width);
	//temp[1] += delta;
	//debugVectFuncLinApprox(sizeV, temp, tempHostInput,
	//							tempHostActGrad, tempHostTarget,
	//							numPixelsPerGroup, numCases,
	//							input.getStride(), tempMatrix[0].getStride(), numColors, sizeH);

	//float sumLA1 =  Sum(tempHostTarget, out_height*out_width);

	//printf("debugVectFunc * s0 %f s1 %f deriv %f\n", sumLA0, sumLA1, (sumLA1-sumLA0)/delta);

//----------


#define ELT_GRAD(SIZE_ARR) \
		if(sizeV == SIZE_ARR){\
			hipFuncSetCacheConfig(reinterpret_cast<const void*>(kVectFuncParamWeightGrad<SIZE_ARR>), hipFuncCachePreferL1);\
			kVectFuncParamWeightGrad<SIZE_ARR><<<blocks, threads, shared_size>>>(actGrad.getDevData(),\
				input.getDevData(), (float**)arrayPtr, numColors, tag_size, numPixelsPerGroup, numCases,\
				input.getStride(), actGrad.getStride(), tempMatrix[0].getStride(), sizeH);};
		ELT_GRAD(1)
		ELT_GRAD(2)
		ELT_GRAD(3)
		ELT_GRAD(4)
		ELT_GRAD(6)
		ELT_GRAD(8)
		ELT_GRAD(12)
		ELT_GRAD(16)
#undef ELT_GRAD

	float sumt = tempMatrix[1].sum();
	printf("kVectFuncParamWeightGrad sum_tag %f \n", sumt);

		cutilCheckMsg("kVectFuncParamWeightGrad: Kernel execution failed");
}