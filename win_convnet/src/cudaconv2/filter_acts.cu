#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cutil_inline.h>
#include <nvmatrix.cuh>
#include <cudaconv2.cuh>

/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of module and B_Y * filtersPerThread
 *
 * images:      (numColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:     (numColors, filterPixels, numFilters) if conv
 *              (numModules, numColors, filterPixels, numFilters) otherwise
 *
 * targets:     (numFilters, numModulesY, numModulesX, numImages)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 *
 * Number of filters per module should be divisible by B_Y * filtersPerThread
 * checkImgBounds indicates whether number of images is divisible by B_X * imgsPerThread
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int numColors,
          bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_color(float* images, float* filters, float* targets,
                                   const int numImages, const int numFilters,
                                   const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                   const int moduleStride,
                                   const int numModulesY, const int numModulesX, const int imgStride,
                                   const float scaleTargets, const float scaleOutputs,
                                   const bool conv) {
    __shared__ float shFilters[B_Y*numColors][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*numColors][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;

	const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = blockIdx.y % blocksPerModule;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;
    images += myImgIdx;
    filters += filtersPerThread * B_Y * blockFilterIdx
             + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx * B_Y * filtersPerThread + threadIdx.y) * numImages * numModulesY * numModulesX
            + myImgIdx;


    float prod[filtersPerThread][imgsPerThread];
	memset(prod, 0, sizeof(prod));

    for (int p = 0; p < filterPixels; p += B_Y) {
        /*
         * Load B_Y pixels from B_Y*filtersPerThread filters
         */
        if (shFilterLoadY < B_Y) {
            #pragma unroll
            for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                if (p + p2 + shFilterLoadY < filterPixels) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[(c * filterPixels + p + p2) * numFilters];
                    }
                } else {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                    }
                }
            }
        }

        /*
         * Load B_Y pixels from B_X*imgsPerThread images
         */
        const int pixIdx = p + threadIdx.y;
        if (pixIdx < filterPixels) {
            const int x = paddingStart + imgLoadModPosX + pixIdx % filterSize;
            const int y = paddingStart + imgLoadModPosY + pixIdx / filterSize;
            if (y >= 0 && y< imgSizeY && x >= 0 && x < imgSizeX) {
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = images[imgStride * (c * imgPixels + y * imgSizeX + x) + i * B_X];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            } else { // Padding
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                    }
                }
            }
        }
        __syncthreads();
        #pragma unroll
        for (int i = 0; i < B_Y*numColors; i++) {
            #pragma unroll
            for(int f = 0; f < filtersPerThread; f++) {
                #pragma unroll
                for(int g = 0; g < imgsPerThread; g++) {
                    prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                }
            }

        }
        __syncthreads();
    }
    
    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleOutputs * prod[f][g];
                }
            }
        }
    }
}

/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of B_Y * filtersPerThread
 *
 * images:      (numImgColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:     (numFilterColors, filterPixels, numFilters) if conv
 *              (numModules, numFilterColors, filterPixels, numFilters) otherwise
 *
 * targets:     (numFilters, numModulesY, numModulesX, numImages)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 * colorCache: how many colors to put into shmem
 *
 * numFilters should be divisible by B_Y * filtersPerThread
 * numImages be divisible by B_X * imgsPerThread
 * numFilterColors should be divisible by colorCache.
 * numImgColors must be even.
 * numFilters must be divisible by numGroups.
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */

#include "tt.h"

template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache,
          bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_sparse(float* images, float* filters, float* targets,
                                       const int numImages, const int numFilters,
                                       const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                       const int moduleStride,
                                       const int numModulesY, const int numModulesX, const int imgStride, const int numImgColors,
                                       const int numGroups, 
                                       const float scaleTargets, const float scaleOutputs,
                                       const bool conv) {
    __shared__ float shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*colorCache][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;
    const int numFilterColors = numImgColors / numGroups;
    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = filtersPerThread * B_Y * (blockIdx.y % blocksPerModule);
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockGroupIdx = blockFilterIdx / numFiltersPerGroup;

    const int numModules = numModulesX * numModulesY;
    const int blockColorIdx = numFilterColors * blockGroupIdx;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = paddingStart + (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = paddingStart + (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;

	const int numFiltersPerBlock = filtersPerThread * B_Y;

//const int numAllFiltersModules = numModules * numFilters
//	dim3(DIVUP(numImages, BX * imgsPerThread), (numModules * numFilters) / (BY * filtersPerThread))
//  const int blocksPerModule = numFilters / (B_Y*filtersPerThread);

	const int& bx = blockIdx.x;
	const int& by = blockIdx.y;

	SPLIT(by, blocksPerModule)
	SPLIT(by_blocksPerModule_x, numModulesX)

	//moduleIdx = by_blocksPerModule_y;
	SPLIT(moduleIdx, moduleStride)

	int oc_, c_, p_, p2_, i_;
	LoopBlock<5> loopBlock;
	loopBlock
	> oc_ < LoopIndex(numFilterColors, colorCache) 
	> c_ < LoopIndex (colorCache, 1) 
	> p_ < LoopIndex (filterPixels, B_Y)
	> p2_ < LoopIndex (B_Y, B_X/filtersPerThread)
	> i_ < LoopIndex (imgsPerThread, 1);


	int oc_l, c_l;
	SplitPos spl_l;

	SBaseIndex<3,1> imgIndex;
	imgIndex << Index(numFilterColors,  blockFilterIdx/numFiltersPerGroup);
/*
	SBaseIndex<3, 1> imgIndex;

	//blockColorIdx = numFilterColors * blockGroupIdx
	//blockGroupIdx = blockFilterIdx / numFiltersPerGroup;
	//numFiltersPerBlock = filtersPerThread * B_Y
	//blockFilterIdx= numFiltersPerBlock * (blockIdx.y % blocksPerModule);
	//by = modueleIdx*blocksPerModule + by_blocksPerModule_x
	//by_blocksPerModule_x = (1...(numModules * numFilters) / (BY * filtersPerThread)) % (numFilters / (B_Y*filtersPerThread))

	imgIndex
	<< Index(numFilterColors,  blockFilterIdx/numFiltersPerGroup)
	>> oc_l << Ref(oc_)
	>> c_l << Ref(c_)

	<< imgSizeY

	<< Index(moduleStride, moduleIdx_moduleStride_y)//center + offset
	>> spl_l << RefSplitY(filterSize, threadIdx.y, p_)
	<< Index(1, paddingStart)

	<< imgSizeX
	
	<< Index(moduleStride, moduleIdx_moduleStride_x)
	<< RefSplitX(p_)
	<< Index(1, paddingStart)

	<< imgStride/B_X

	<< Index(imgsPerThread , bx)
	<< Ref(i_)
	<< B_X
	<< Index(1, threadIdx.x);

//------------------------------------
	int p_l, p2_l;
	BaseIndex<4> filterIndex;
	filterIndex <<Index(0,0)

	>> oc_l << Ref(oc_)
	>> c_l << Ref(c_)

	<< filterPixels

	>> p_l << Ref(p_)
	>> p2_l << Ref(p2_)
	<< shFilterLoadY

	<< numFilters

	<< blockFilterIdx // numFiltersPerBlock * (1..numFilters / numFiltersPerBlock) X numModules

	<< shFilterLoadX; //(1..numFiltersPerBlock) X B_X/numFilterPerThread

  // shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
//  shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];

	BaseIndex<2> shFilterIndex;
	
	shFilterIndex
	>> c_l << Ref(c_)
	<< B_Y
	>> p2_l << Ref(p2_)
	<< shFilterLoadY
	<< numFiltersPerBlock
	<< shFilterLoadX;

*/
/*


//--------------------------------
	Index shFilterIndex;
	shFilterIndex
	< SIndex(B_Y, c_filter)<<B_Y * filtersPerThread //parallel->seq color

	< SIndex(B_X/filtersPerThread,  p2)<<B_Y * filtersPerThread // p2 ->B_Y pixel, make B_Y pixels parallel->seq
	< Index(1, tidx /(B_Y * filtersPerThread))<<B_Y * filtersPerThread //shFilterLoadY pixel

	< Index(1, tidx % (B_Y * filtersPerThread)); //shFilterLoadX filter, parallel

//--------------------------------


	Index tagIndex;
	tagIndex
	< Index(numImages * numModules,  filtersPerThread * B_Y*SplitX(blockIdx.y, blocksPerModule))
	< SIndex(B_Y*numImages * numModules, f)
	< Index(numImages * numModules, threadIdx.y)
	< Index(numImages, blockIdx.y / blocksPerModule) 
	< Index(B_X * imgsPerThread , blockIdx.x * B_X)//myImgIdx
	< SIndex(B_X, g)
	< Index(1, threadIdx.x);//myImgIdx 


*/
    images += blockColorIdx * imgPixels * imgStride + myImgIdx;
    filters +=blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numFilterColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx + threadIdx.y) * numImages * numModules
            + myImgIdx;

    float prod[filtersPerThread][imgsPerThread];
	memset(prod, 0, sizeof(prod));

//    __shared__ int imgPos[]
    for (int oc = 0; oc < numFilterColors; oc += colorCache) { // oc stands for outer color (loop)
        for (int p = 0; p < filterPixels; p += B_Y) {
            /*
             * Load B_Y pixels from B_Y*filtersPerThread filters
             */
            if (shFilterLoadY < B_Y) {
                #pragma unroll
                for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                    if (p + p2 + shFilterLoadY < filterPixels) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                        }
                    }
                }
            }

            /*
             * Load B_Y pixels from B_X*imgsPerThread images
             */
            const int pixIdx = p + threadIdx.y;
            if (pixIdx < filterPixels) {
                const int x = imgLoadModPosX + pixIdx % filterSize;
                const int y = imgLoadModPosY + pixIdx / filterSize;
                if (y >= 0 && y < imgSizeY && x >= 0 && x < imgSizeX) {
                    float* m = &images[imgStride * (oc * imgPixels + y * imgSizeX + x)];
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = m[c * imgStride * imgPixels + i * B_X];
                            }
                        } else {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                            }
                        }
                    }
                } else { // Padding
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            }
            __syncthreads();
            #pragma unroll
            for (int i = 0; i < B_Y*colorCache; i++) {
                #pragma unroll
                for(int f = 0; f < filtersPerThread; f++) {
                    #pragma unroll
                    for(int g = 0; g < imgsPerThread; g++) {
                        prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                    }
                }

            }
            __syncthreads();
        }
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModules] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleOutputs * prod[f][g];
                }
            }
        }
    }
}


/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of B_Y * filtersPerThread
 *
 * images:          (numImgColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:         (numFilterColors, filterPixels, numFilters) if conv
 *                  (numModules, numFilterColors, filterPixels, numFilters) otherwise
 *
 * targets:         (numFilters, numModulesY, numModulesX, numImages)
 * colorIndices:    (numGroups, numFiltercolors)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 * colorCache: how many colors to put into shmem
 *
 * numFilters should be divisible by B_Y * filtersPerThread
 * numImages be divisible by B_X * imgsPerThread
 * numFilterColors should be divisible by colorCache.
 * numImgColors must be even.
 * numFilters must be divisible by numGroups.
 *
 * The imgSize here is the size of the actual image without the padding.
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_sparse_random(float* images, float* filters, float* targets, int* colorIndices,
                                             const int numImages, const int numFilters,
                                             const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                             const int moduleStride,
                                             const int numModulesY, const int numModulesX, const int imgStride,
                                             /*const int numImgColors,*/ const int numFilterColors, const int numGroups, 
                                             const float scaleTargets, const float scaleOutputs,
                                             const bool conv) {
    __shared__ float shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*colorCache][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    __shared__ int shColors[colorCache];
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;
//    const int numFilterColors = numImgColors / numGroups;
    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = filtersPerThread * B_Y * (blockIdx.y % blocksPerModule);
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockGroupIdx = blockFilterIdx / numFiltersPerGroup;

    const int numModules = numModulesY * numModulesX;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = paddingStart + (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = paddingStart + (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;

    images += myImgIdx;
    filters +=blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numFilterColors * filterPixels * numFilters;
    }
    
    targets += moduleIdx * numImages
            + (blockFilterIdx + threadIdx.y) * numImages * numModules
            + myImgIdx;
    colorIndices += blockGroupIdx * numFilterColors;

    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }
//    __shared__ int imgPos[]
    for (int oc = 0; oc < numFilterColors; oc += colorCache) { // oc stands for outer color (loop)
        
        // Kinda wasteful here but...shouldn't matter
        if (tidx < colorCache) {
            shColors[tidx] = colorIndices[oc + tidx] * imgStride * imgPixels;
        }
        __syncthreads();
        for (int p = 0; p < filterPixels; p += B_Y) {
            /*
             * Load B_Y pixels from B_Y*filtersPerThread filters
             */
            if (shFilterLoadY < B_Y) {
                #pragma unroll
                for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                    if (p + p2 + shFilterLoadY < filterPixels) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                        }
                    }
                }
            }

            /*
             * Load B_Y pixels from B_X*imgsPerThread images
             */
            const int pixIdx = p + threadIdx.y;
            if (pixIdx < filterPixels) {
                const int x = imgLoadModPosX + pixIdx % filterSize;
                const int y = imgLoadModPosY + pixIdx / filterSize;
                if (y >= 0 && y < imgSizeY && x >= 0 && x < imgSizeX) {
                    float* m = &images[imgStride * (y * imgSizeX + x)];
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = m[shColors[c] + i * B_X];
                            }
                        } else {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                            }
                        }
                    }
                } else { // Padding
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            }
            __syncthreads();
            #pragma unroll
            for (int i = 0; i < B_Y*colorCache; i++) {
                #pragma unroll
                for(int f = 0; f < filtersPerThread; f++) {
                    #pragma unroll
                    for(int g = 0; g < imgsPerThread; g++) {
                        prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                    }
                }

            }
            __syncthreads();
        }
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModules] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleOutputs * prod[f][g];
                }
            }
        }
    }
}

// numbers of threads per block
//B_Y
#define FILTER_THREADS 4
//B_X
#define IMAGE_THREADS 32
/*
 * images:      (numImgColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:     (numFilterColors, filterPixels, numFilters)             if conv
 *              (numModules, numFilterColors, filterPixels, numFilters) otherwise
 *
 * targets:     (numFilters, numModules, numImages)
 * 
 * Note: all of these convolution routines are optimized for the case when
 * the number of images (i.e. the minibatch size) is a multiple of 128. 
 * Other batch sizes will work, but but I made no attempt whatsoever
 * to make them work fast. 
 */
 void _filterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                   int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                   int numImgColors, int numGroups,
                   float scaleTargets, float scaleOutput, bool conv) {
    int numFilterColors = numImgColors / numGroups;      
    int numFilters = filters.getNumCols();
    int numModules = numModulesY * numModulesX;
    int numImages = images.getNumCols();
    int imgPixels = images.getNumRows()/numImgColors;
    int imgSizeX = imgPixels / imgSizeY;
    int filterModuleMult = conv ? 1 : numModules;
    
    assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
    assert(numGroups == 1 || numFilterColors % 2 == 0);
    assert(numFilters % (16 * numGroups) == 0);
    assert(numImgColors % numGroups == 0);
    assert(images.getNumRows() == imgPixels * numImgColors);
    assert(imgSizeY * imgSizeX == imgPixels);
    int numFiltersPerGroup = numFilters / numGroups;

    int imgStride = images.getStride(); // images does not need to be a contiguous matrix

    int filterPixels = filters.getNumRows() / (filterModuleMult * numFilterColors);
    int filterSize = int(sqrt(filterPixels));
    assert(filterSize * filterSize == filterPixels);
    assert(filters.getNumRows() == filterModuleMult * numFilterColors * filterPixels);

    // These routines don't handle the case when only part of the image is visited in the convolution
    assert(paddingStart <= 0);
    assert(paddingStart + (numModulesX-1)*moduleStride + filterSize >= imgSizeX);
    assert(paddingStart + (numModulesY-1)*moduleStride + filterSize >= imgSizeY);
    assert(moduleStride <= filterSize);
    
    assert(!images.isTrans());
    assert(!filters.isTrans());
    assert(!targets.isTrans());

    assert(filters.isContiguous());
    assert(targets.isContiguous());
    int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;

//	numFiltersPerGroup % 32 == 0 ? filtersPerThread = 8
//		: filtersPerThread = 4


    dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, IMAGE_THREADS * imgsPerThread), (numModules * numFilters) / (FILTER_THREADS * 8))
                                               : dim3(DIVUP(numImages, IMAGE_THREADS * imgsPerThread), (numModules * numFilters) / (FILTER_THREADS * 4));
    dim3 threads(IMAGE_THREADS, FILTER_THREADS);
    bool checkImgBounds = numImages % (IMAGE_THREADS*imgsPerThread) != 0;
    if (scaleTargets == 0) {
        targets.resize(numFilters * numModules, numImages);
    } else {
        assert(targets.getNumRows() == numFilters * numModules);
        assert(targets.getNumCols() == numImages);
    }

    if (imgsPerThread == 4) {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 8, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 8, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 4, 4, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 4, 4, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    } else if (imgsPerThread == 2) {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 8, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 8, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 2, 4, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 2, 4, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }    
    } else {
        if (numImgColors <= 3) {
            assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 1, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 1, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 3, false, true >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 3, false, false >, hipFuncCachePreferShared);
                             filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 1, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 1, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 3, true, true >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 8, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 8, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< FILTER_THREADS), IMAGE_THREADS, 1, 4, 3, true, false >, hipFuncCachePreferShared);
                            filterActs_YxX_color < FILTER_THREADS, IMAGE_THREADS, 1, 4, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), IMAGE_THREADS, 1, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    }
    
    cutilCheckMsg("filterActs: kernel execution failed");
}

void convFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numGroups) {
    convFilterActs(images, filters, targets, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1);
}

void convFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                   int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                   int numImgColors, int numGroups,
                   float scaleTargets, float scaleOutput) {
     _filterActs(images, filters, targets, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput, true);
}

void localFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numGroups) {
    localFilterActs(images, filters, targets, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1);
}

void localFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                   int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                   int numImgColors, int numGroups,
                   float scaleTargets, float scaleOutput) {
     _filterActs(images, filters, targets, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput, false);
}

/*
 * images:          (numImgColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:         (numFilterColors, filterPixels, numFilters)             if conv
 *                  (numModules, numFilterColors, filterPixels, numFilters) otherwise
 *
 * targets:         (numFilters, numModulesY, numModulesX, numImages)
 * colorIndices:    (numGroups, numFilterColors)
 * 
 * Note: all of these convolution routines are optimized for the case when
 * the number of images (i.e. the minibatch size) is a multiple of 128. 
 * Other batch sizes will work, but but I made no attempt whatsoever
 * to make them work fast. 
 */
void _filterActsSparse(NVMatrix& images, NVMatrix& filters, NVMatrix& targets, int* dColorIndices,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numFilterColors, int numGroups,
                          float scaleTargets, float scaleOutput, bool conv) {
    int numFilters = filters.getNumCols();
    int numModules = numModulesY * numModulesX;
    int numImages = images.getNumCols();
    int imgPixels = images.getNumRows() / numImgColors;
    int imgSizeX = imgPixels / imgSizeY;
    int filterModuleMult = conv ? 1 : numModules;
    
    assert(numGroups > 1);
    assert(numImgColors % numFilterColors == 0);
    assert((numFilterColors * numGroups) % numImgColors == 0);
    assert(numFilters % (16 * numGroups) == 0);
    assert(numFilterColors % 2 == 0);
    
    assert(imgSizeY * imgSizeX == imgPixels);
    assert(images.getNumRows() == imgPixels * numImgColors);
    int numFiltersPerGroup = numFilters / numGroups;

    int imgStride = images.getStride(); // images does not need to be a contiguous matrix

    int filterPixels = filters.getNumRows() / (filterModuleMult * numFilterColors);
    int filterSize = int(sqrt(filterPixels));
    assert(filterSize * filterSize == filterPixels);
    assert(filters.getNumRows() == filterModuleMult * numFilterColors * filterPixels);

    // These routines don't handle the case when only part of the image is visited in the convolution
    assert(paddingStart <= 0);
    assert(paddingStart + (numModulesX-1) * moduleStride + filterSize >= imgSizeX);
    assert(paddingStart + (numModulesY-1) * moduleStride + filterSize >= imgSizeY);
    assert(moduleStride <= filterSize);
    
    assert(!images.isTrans());
    assert(!filters.isTrans());
    assert(!targets.isTrans());

    assert(filters.isContiguous());
    assert(targets.isContiguous());
    int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;

    dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, IMAGE_THREADS * imgsPerThread), (numModules * numFilters) / (FILTER_THREADS * 8))
                                               : dim3(DIVUP(numImages, IMAGE_THREADS * imgsPerThread), (numModules * numFilters) / (FILTER_THREADS * 4));
    dim3 threads(IMAGE_THREADS, FILTER_THREADS);
    bool checkImgBounds = numImages % (IMAGE_THREADS*imgsPerThread) != 0;
    if (scaleTargets == 0) {
        targets.resize(numFilters * numModules, numImages);
    } else {
        assert(targets.getNumRows() == numFilters * numModules);
        assert(targets.getNumCols() == numImages);
    }
    
    if (imgsPerThread == 4) {
        if (scaleTargets == 0) { // don't scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        } else { // do scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 4, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        }
    } else if (imgsPerThread == 2) {
        if (scaleTargets == 0) { // don't scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        } else { // do scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 2, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 2, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        }
    } else {
        if (scaleTargets == 0) { // don't scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        } else { // do scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse_random< FILTER_THREADS), IMAGE_THREADS, 1, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse_random < FILTER_THREADS, IMAGE_THREADS, 1, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(), dColorIndices,
                                numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numFilterColors, numGroups, scaleTargets, scaleOutput, conv);
                }
            }
        }
    }
    
    cutilCheckMsg("filterActsSparse: kernel execution failed");
}

void convFilterActsSparse(NVMatrix& images, NVMatrix& filters, NVMatrix& targets, int* dColorIndices,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numFilterColors, int numGroups,
                          float scaleTargets, float scaleOutput) { 
    _filterActsSparse(images, filters, targets, dColorIndices, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride,
                      numImgColors,  numFilterColors, numGroups, scaleTargets, scaleOutput, true);
}

void convFilterActsSparse(NVMatrix& images, NVMatrix& filters, NVMatrix& targets, int* dColorIndices,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numFilterColors, int numGroups) {
    convFilterActsSparse(images, filters, targets, dColorIndices, imgSizeY, numModulesY, numModulesX, paddingStart,
                         moduleStride, numImgColors, numFilterColors, numGroups, 0, 1);
}

void localFilterActsSparse(NVMatrix& images, NVMatrix& filters, NVMatrix& targets, int* dColorIndices,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numFilterColors, int numGroups,
                          float scaleTargets, float scaleOutput) { 
    _filterActsSparse(images, filters, targets, dColorIndices, imgSizeY, numModulesY, numModulesX, paddingStart, moduleStride,
                      numImgColors,  numFilterColors, numGroups, scaleTargets, scaleOutput, false);
}

void localFilterActsSparse(NVMatrix& images, NVMatrix& filters, NVMatrix& targets, int* dColorIndices,
                          int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numFilterColors, int numGroups) {
    localFilterActsSparse(images, filters, targets, dColorIndices, imgSizeY, numModulesY, numModulesX, paddingStart,
                         moduleStride, numImgColors, numFilterColors, numGroups, 0, 1);
}
