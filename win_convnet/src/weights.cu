#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>

bool Weights::_autoCopyToGPU = false;

int AUX_STORAGE = 32;

void Weights::shrink(float lambda)
{
	if (_wc > 0) {
		_weights->shrink(lambda);			
	}
};

extern int rnd_aux;

void Weights::procAux() {

	if(!_active_aux)
		return;

    assert(_onGPU);

	if(!_weightsGrad->isSameDims(getAuxSum()));
		getAuxSum().resize(*_weightsGrad);

	if(_aux_filled == 0)
	{
		zeroAux();
	}

	if(_aux_filled >= 0)
		getAuxSum().add(*_weightsGrad, 1.);

	if(_aux_filled >= _aux_store_size)
	{
		assert(getAuxSum().isSameDims(getAuxUpdate()));
		getAuxSum().add(getAuxUpdate(), -1.);//remove
	}

	if(!_weightsGrad->isSameDims(getAuxUpdate()));
		getAuxUpdate().resize(*_weightsGrad);

	_weightsGrad->copy(getAuxUpdate());

	_aux_filled = min(_aux_filled+1, _aux_store_size);
	_aux_update = (_aux_update+1)%_aux_store_size;

}

void Weights::stepAuxInd()
{
	_aux_update = (_aux_update+1)%_aux_store_size;
	_aux_filled = min(_aux_filled+1, _aux_store_size);
}


void Weights::zeroAux() {
	if(_active_aux)
		getAuxSum().apply(NVMatrixOps::Zero());
}

void Weights::zeroAux(int ind) {
	if(_active_aux)
		getAux(ind).apply(NVMatrixOps::Zero());
}

void Weights::rollback(float reduceScale) 
{
    assert(_onGPU);
	_weights->add(*_weightsInc, reduceScale-1);
}

// Scale your gradient by epsW / numCases!
void Weights::update(bool useAux) {
    // Only true owner of weights updates
    if (_srcWeights == NULL && _epsW > 0) {

        assert(_onGPU);
        if (_useGrad) {
//rmsprop
			float scaleGrad = 1;
			{
				float norm2 =  _weightsGrad->norm2();
				int size = _weightsGrad->getNumElements();	
				
				_norms_size = 128;
				while(_norms2.size() < _norms_size)
					_norms2.push_back(0);

				if(_epsW != _epsWprev)
					_norms_filled = 0;
				
				if(_norms_filled == _norms_size)
				{
					assert(_rmsW > 0 && _rmsW < .01);
					scaleGrad = _epsW/_epsWinit*_rmsW/getNormL2Avg();
				}

				getNorm2Update() = norm2;

				_norms_filled = min(_norms_filled+1, _norms_size);
				_norms_update = (_norms_update+1)%_norms_size;
				_epsWprev = _epsW;
			}
//rmsprop end


			_weightsInc->add(*_weightsGrad, _mom, scaleGrad);
	
        }

        if (_wc > 0) {
            //_weightsInc->addSignReg(*_weights, -_wc * _epsW);	
			_weightsInc->add(*_weights, -_wc * _epsW);				
        }

		//nesterov
		//if(_active_aux && useAux )
		//{
		//	getAux(0).add(*_weightsInc);
		//	getAux(0).add(*_weightsInc, 1, _mom, *_weights);
		//}
		//else	        

		_weights->add(*_weightsInc);

		_numUpdates = 0;

		if(_renorm > 0)
		{

			float norm2 =  _weights->norm2();
			int size = _weights->getNumElements();	
			float layerNorm = sqrtf(norm2/size);

			if(layerNorm > _renorm)
			{	
				float renormScale = _renorm/layerNorm;
				_weights->scale(renormScale);
			}
		}

    }
}

float Weights::getNormL2Avg()
{

	float l2 = 0;
	for(int i = 0; i < _norms_filled; i++)
		l2 += _norms2[i];

	float ninv = 0;
	if(_norms_filled > 0)ninv = 1./_norms_filled;
	return sqrt(l2*ninv);
}


void Weights::copyToCPU() {

    if (_srcWeights == NULL) {
        assert(_onGPU);
        _weights->copyToHost(*_hWeights);
        _weightsInc->copyToHost(*_hWeightsInc);
//bregman
		//if(_active_aux && _hAux_weights)
		//{
		//	_aux_weights[_aux_update].copyToHost(*_hAux_weights);
		//}
    }
}

void Weights::initAux()
{
	_aux_filled = 0;
	_aux_update = 0;

	for(int i = 0; i < _full_store_size; i++)
		_aux_weights.push_back(NVMatrix());

	//if(!_weightsInc->isSameDims(getAux(0)))
	//	getAux(0).resize(*_weightsInc);

	//_weightsInc->copy(getAux(0));


	//if(!_weights->isSameDims(getAux(0)))
	//	getAux(0).resize(*_weights);

	//_weights->copy(getAux(0));

	//_aux_weights[0].copyFromHost(*_hAux_weights, true);

	//for(int i = 1; i < _full_store_size; i++)
	//{

	//	_aux_weights[i].resize(_aux_weights[0]);
	//	_aux_weights[i].apply(NVMatrixOps::Zero());
	//}

}

void Weights::CopyGradToAux()
{
	assert(_useGrad);
	getAuxUpdate().resize(*_weightsGrad);
	_weightsGrad->copy(getAuxUpdate());
};

void Weights::setAuxUpdateInd(int updInd)
{
	_aux_update = updInd;
}

int Weights::getAuxUpdateInd()
{
	return _aux_update;
}

void Weights::copyToGPU() {

    if (_srcWeights == NULL) {

        _weights = new NVMatrix();
        _weightsInc = new NVMatrix();
        _weights->copyFromHost(*_hWeights, true);
        _weightsInc->copyFromHost(*_hWeightsInc, true);
        _weightsGrad = _useGrad ? new NVMatrix() : NULL;

	    _onGPU = true;

		//bregman
		if(_active_aux)
			initAux();

    } else {
        _weights = _srcWeights->_weights;
        _weightsInc = _srcWeights->_weightsInc;
        _weightsGrad = _srcWeights->_weightsGrad;

	    _onGPU = true;

		//bregman
		if(_active_aux)
			initAux();
    }
}
    