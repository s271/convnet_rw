#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <weights.cuh>

bool Weights::_autoCopyToGPU = false;

int AUX_STORAGE = 2;
//#define USE_PREC

void Weights::shrink(float lambda)
{
	if (_wc > 0) {
		_weights->shrink(lambda);			
	}
};

extern int rnd_aux;

void Weights::procAux() {

	if(!_active_aux)
		return;

    assert(_onGPU);

	//if(!_weightsGrad->isSameDims(getAuxSum()));
	//	getAuxSum().resize(*_weightsGrad);

	//if(_aux_filled == 0)
	//{
	//	zeroAux();
	//}

	//if(_aux_filled >= 0)
	//	getAuxSum().add(*_weightsGrad, 1.);

	//if(_aux_filled >= _aux_store_size)
	//{
	//	assert(getAuxSum().isSameDims(getAuxUpdate()));
	//	getAuxSum().add(getAuxUpdate(), -1.);//remove
	//}

	//if(!_weightsGrad->isSameDims(getAuxUpdate()));
	//	getAuxUpdate().resize(*_weightsGrad);

	//_weightsGrad->copy(getAuxUpdate());

	//_aux_filled = min(_aux_filled+1, _aux_store_size);
	//_aux_update = (_aux_update+1)%_aux_store_size;

}

void Weights::stepAuxInd()
{
	//_aux_update = (_aux_update+1)%_aux_store_size;
	//_aux_filled = min(_aux_filled+1, _aux_store_size);
}


void Weights::zeroAux() {
	if(_active_aux)
		getAuxSum().apply(NVMatrixOps::Zero());
}

void Weights::zeroAux(int ind) {
	if(_active_aux)
		getAux(ind).apply(NVMatrixOps::Zero());
}

#define BASE 0
#define PREV_GRAD 1
void Weights::rollback(float reduceScale) 
{
#ifdef USE_PREC
	getAuxSum().add(*_weightsInc, reduceScale-1);
#else	
	_weights->add(*_weightsInc, reduceScale-1);
#endif
}

// Scale your gradient by epsW / numCases!
void Weights::update(bool useAux) {
    // Only true owner of weights updates
    if (_srcWeights == NULL && _epsW > 0) {

        assert(_onGPU);
        if (_useGrad) {

			float scaleGrad = 1;

			//if(_weightsGrad->isSameDims(getAux(PREV_GRAD)))
			//{
			//	//float g2 = _weightsGrad->norm2();
			//	//float g2prev = getAux(PREV_GRAD).norm2();
			//	float dot =   getAux(PREV_GRAD).dotProduct(*_weightsGrad);
			//	//printf("1e9x g2 %f dot/g2prev %f g2/g2prev %f\n", 1e9*g2, dot/g2prev, g2/g2prev);
			//	if(dot < 0)
			//		scaleGrad *= .2;
			//}	

			_weightsInc->add(*_weightsGrad, _mom, scaleGrad);

			//if(!_weightsGrad->isSameDims(getAux(PREV_GRAD)));
			//	getAux(PREV_GRAD).resize(*_weightsGrad);
			//_weightsGrad->copy(getAux(PREV_GRAD));

        }

        if (_wc > 0) {
            //_weightsInc->addSignReg(*_weights, -_wc * _epsW);	
			_weightsInc->add(*_weights, -_wc * _epsW);				
        }
  
#ifdef USE_PREC
		if(!_weightsInc->isSameDims(getAuxSum()))
		{
			getAuxSum().resize(*_weightsInc);
			getAux(BASE).resize(*_weights);
			zeroAux();
			_weights->copy(getAux(BASE));
			_aux_filled = 0;
		}

		getAuxSum().add(*_weightsInc);
		getAuxSum().add(getAux(BASE), *_weights);
		_aux_filled++;
#else
		_weights->add(*_weightsInc);
#endif
		_numUpdates = 0;

		if(_renorm > 0)
		{

			float norm2 =  _weights->norm2();
			int size = _weights->getNumElements();	
			float layerNorm = sqrtf(norm2/size);

			if(layerNorm > _renorm)
			{	
				float renormScale = _renorm/layerNorm;
				_weights->scale(renormScale);
				getAuxSum().scale(renormScale);
				getAux(BASE).scale(renormScale);
			}
		}
#ifdef USE_PREC
		if(_aux_filled >= 64)
		{
			_weights->copy(getAux(BASE));
			zeroAux();
			_aux_filled = 0;
		}
#endif
    }
}

float Weights::getNormL2Avg()
{

	float l2 = 0;
	for(int i = 0; i < _norms_filled; i++)
		l2 += _norms2[i];

	float ninv = 0;
	if(_norms_filled > 0)ninv = 1./_norms_filled;
	return sqrt(l2*ninv);
}


void Weights::copyToCPU() {

    if (_srcWeights == NULL) {
        assert(_onGPU);
        _weights->copyToHost(*_hWeights);
        _weightsInc->copyToHost(*_hWeightsInc);
//bregman
		//if(_active_aux && _hAux_weights)
		//{
		//	_aux_weights[_aux_update].copyToHost(*_hAux_weights);
		//}
    }
}

void Weights::initAux()
{
	_aux_filled = 0;
	_aux_update = 0;

	for(int i = 0; i < _full_store_size; i++)
		_aux_weights.push_back(NVMatrix());

	//if(!_weightsInc->isSameDims(getAux(0)))
	//	getAux(0).resize(*_weightsInc);

	//_weightsInc->copy(getAux(0));


	//if(!_weights->isSameDims(getAux(0)))
	//	getAux(0).resize(*_weights);

	//_weights->copy(getAux(0));

	//_aux_weights[0].copyFromHost(*_hAux_weights, true);

	//for(int i = 1; i < _full_store_size; i++)
	//{

	//	_aux_weights[i].resize(_aux_weights[0]);
	//	_aux_weights[i].apply(NVMatrixOps::Zero());
	//}

}

void Weights::CopyGradToAux()
{
	assert(_useGrad);
	getAuxUpdate().resize(*_weightsGrad);
	_weightsGrad->copy(getAuxUpdate());
};

void Weights::setAuxUpdateInd(int updInd)
{
	_aux_update = updInd;
}

int Weights::getAuxUpdateInd()
{
	return _aux_update;
}

void Weights::copyToGPU() {

    if (_srcWeights == NULL) {

        _weights = new NVMatrix();
        _weightsInc = new NVMatrix();
        _weights->copyFromHost(*_hWeights, true);
        _weightsInc->copyFromHost(*_hWeightsInc, true);
        _weightsGrad = _useGrad ? new NVMatrix() : NULL;

	    _onGPU = true;

		//bregman
		if(_active_aux)
			initAux();

    } else {
        _weights = _srcWeights->_weights;
        _weightsInc = _srcWeights->_weightsInc;
        _weightsGrad = _srcWeights->_weightsGrad;

	    _onGPU = true;

		//bregman
		if(_active_aux)
			initAux();
    }
}
    